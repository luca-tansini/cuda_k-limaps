#include "MoorePenrose.cu"
#include <stdio.h>

int main(int argc, char **argv){

    int n,m,i;

    if(argc != 3){
        printf("usage: JacobiMoorePenroseTest n m\n");
        exit(-1);
    }

    n = atoi(argv[1]);
    m = atoi(argv[2]);

    srand(time(NULL));
    double *A;
    CHECK(hipHostMalloc(&A, n*m*sizeof(double)));

    for(i=0; i<n*m; i++)
        A[i] = rand()/(double)RAND_MAX;

    double *d_A,*d_Apinv;
    CHECK(hipMalloc(&d_A, n*m*sizeof(double)));
    CHECK(hipMalloc(&d_Apinv, n*m*sizeof(double)));

    CHECK(hipMemcpy(d_A, A, n*m*sizeof(double), hipMemcpyHostToDevice));

    //call JacobiMoorePenroseInverse
    double t = seconds();
    JacobiMoorePenroseInverse(d_A, n, m, d_Apinv);
    printf("elapsed time: %fs\n",seconds() - t);

    //Check result
    if(CheckPseudoinverse(d_A, n, m, d_Apinv)){
        printf("OK\n");
    }
    else{
        printf("NOPE\n");
    }

    return 0;
}
