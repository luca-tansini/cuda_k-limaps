#include "MoorePenrose.cu"
#include <stdio.h>

/*
Questo programma testa l'algoritmo per il calcolo della pseudoinversa di MoorePenrose che utilizza il metodo di Jacobi.
*/
int main(int argc, char **argv){

    int n,m,i;

    if(argc != 3){
        printf("usage: JacobiMoorePenroseTest n m\n");
        exit(-1);
    }

    n = atoi(argv[1]);
    m = atoi(argv[2]);

    srand(time(NULL));
    double *A;
    CHECK(hipHostMalloc(&A, n*m*sizeof(double)));

    for(i=0; i<n*m; i++)
        A[i] = rand()/(double)RAND_MAX;

    double *d_A,*d_Apinv;
    CHECK(hipMalloc(&d_A, n*m*sizeof(double)));
    CHECK(hipMalloc(&d_Apinv, n*m*sizeof(double)));

    CHECK(hipMemcpy(d_A, A, n*m*sizeof(double), hipMemcpyHostToDevice));

    //Chiama JacobiMoorePenroseInverse
    double t = seconds();
    JacobiMoorePenroseInverse(d_A, n, m, d_Apinv);
    printf("elapsed time: %fs\n",seconds() - t);

    //Controllo del risultato
    if(CheckPseudoinverse(d_A, n, m, d_Apinv)){
        printf("OK\n");
    }
    else{
        printf("NOPE\n");
    }

    return 0;
}
