#include "hip/hip_runtime.h"
#include "vectorUtility.cu"

#define BLOCK_SIZE 256

/*
Function generating the dictionary
The values are extracted from a normal distribution (mean 0, stdev 1.0)
Then each column is normalized, dividing each element by the column's norm
*/
void createDict(double *D, int n, int m){

    int blocksperdict = ceil(n*m*1.0/BLOCK_SIZE);
    int blockspercol = ceil(n*1.0/BLOCK_SIZE);

    srand(time(NULL));
    int seed = rand();

    hiprandState *devStates;
    CHECK(hipMalloc((void **)&devStates, blocksperdict*BLOCK_SIZE*sizeof(hiprandState)));

    normfill<<<blocksperdict,BLOCK_SIZE>>>(D, n*m, devStates, seed);
    CHECK(hipDeviceSynchronize());

    double *tmpcol;
    CHECK(hipMalloc(&tmpcol, blockspercol*BLOCK_SIZE*sizeof(double)));

    for(int i=0; i<m; i++){

        //Copia per il calcolo della norma
        CHECK(hipMemcpy(tmpcol, &D[i*n], n*sizeof(double), hipMemcpyDeviceToDevice));

        //Calcola norma
        double norm = vectorNorm(tmpcol,n);

        //CHIAMA KERNEL CHE DIVIDE OGNI ELEMENTO PER LA NORMA
        if(norm != 0){
            divide<<<blockspercol,BLOCK_SIZE>>>(&D[i*n], norm, n);
            CHECK(hipDeviceSynchronize());
        }
    }
}

/*
Function generating the k-sparse vector alpha
The k values are extracted from a normal distribution (mean 0, stdev 1.0)
*/
void generateAlpha(double *alpha, int m, int k){

    int blocksperk = ceil(k*1.0/BLOCK_SIZE);

    srand(time(NULL));
    int seed = rand();

    hiprandState *devStates;
    CHECK(hipMalloc(&devStates, blocksperk*BLOCK_SIZE*sizeof(hiprandState)));

    double *d_kvalues,h_kvalues[k];
    CHECK(hipMalloc(&d_kvalues, k*sizeof(double)));

    normfill<<<blocksperk,BLOCK_SIZE>>>(d_kvalues, k, devStates, seed);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(h_kvalues, d_kvalues, k*sizeof(double), hipMemcpyDeviceToHost));

    double h_alpha[m];
    memset(h_alpha, 0, m*sizeof(double));
    for(int i=0; i<k; i++){
        int idx = rand()%m;
        if(h_alpha[idx] != 0)
            i--;
        else
            h_alpha[idx] = h_kvalues[i];
    }

    CHECK(hipMemcpy(alpha, h_alpha, m * sizeof(double), hipMemcpyHostToDevice));

}
