#include "hip/hip_runtime.h"
#include "vectorUtility.cu"

#define BLOCK_SIZE 256

/*
Funzione che genera il dizionario D
I valori sono estratti dalla distribuzione gaussiana (media 0, stdev 1.0)
Ogni colonna è normalizzata (dividendo ogni elemento per la norma della colonna)
*/
void createDict(double *D, int n, int m){

    int blocksperdict = ceil(n*m*1.0/BLOCK_SIZE);
    int blockspercol = ceil(n*1.0/BLOCK_SIZE);

    srand(time(NULL));
    int seed = rand();

    hiprandState *devStates;
    CHECK(hipMalloc((void **)&devStates, blocksperdict*BLOCK_SIZE*sizeof(hiprandState)));

    normfill<<<blocksperdict,BLOCK_SIZE>>>(D, n*m, devStates, seed);
    CHECK(hipDeviceSynchronize());

    double *tmpcol;
    CHECK(hipMalloc(&tmpcol, blockspercol*BLOCK_SIZE*sizeof(double)));

    for(int i=0; i<m; i++){

        //Copia per il calcolo della norma
        CHECK(hipMemcpy(tmpcol, &D[i*n], n*sizeof(double), hipMemcpyDeviceToDevice));

        //Calcola norma
        double norm = vectorNorm(tmpcol,n);

        //Chiama kernel che divide ogni elemento della colonna per la norma
        if(norm != 0){
            divide<<<blockspercol,BLOCK_SIZE>>>(&D[i*n], norm, n);
            CHECK(hipDeviceSynchronize());
        }
    }
}

/*
Funzione che genera il vettore k-sparso alpha
I k valori sono estratti da una distribuzione normale (media 0, stdev 1.0)
*/
void generateAlpha(double *alpha, int m, int k){

    int blocksperk = ceil(k*1.0/BLOCK_SIZE);

    srand(time(NULL));
    int seed = rand();

    hiprandState *devStates;
    CHECK(hipMalloc(&devStates, blocksperk*BLOCK_SIZE*sizeof(hiprandState)));

    double *d_kvalues,h_kvalues[k];
    CHECK(hipMalloc(&d_kvalues, k*sizeof(double)));

    normfill<<<blocksperk,BLOCK_SIZE>>>(d_kvalues, k, devStates, seed);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(h_kvalues, d_kvalues, k*sizeof(double), hipMemcpyDeviceToHost));

    double h_alpha[m];
    memset(h_alpha, 0, m*sizeof(double));
    for(int i=0; i<k; i++){
        int idx = rand()%m;
        if(h_alpha[idx] != 0)
            i--;
        else
            h_alpha[idx] = h_kvalues[i];
    }

    CHECK(hipMemcpy(alpha, h_alpha, m * sizeof(double), hipMemcpyHostToDevice));

}
