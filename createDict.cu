#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "vectorUtility.cu"

#define BLOCK_SIZE 256

__global__ void normfill(double *D, int len, hiprandState *states, int seed){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < len){
        hiprand_init(tid*seed+seed, 0, 0, &states[tid]);
        D[tid] = hiprand_uniform_double(&states[tid]);
    }

}

__global__ void divide(double *v, double x, int len){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < len)
        v[tid] /= x;

}

//Function generating the dictionary
//The values are extracted from a normal distribution (mean 0, stdev 1.0)
//Then each column is normalized, dividing each element by the column's norm
void createDict(double *D, int n, int m){

    int blocksperdict = ceil(n*m*1.0/BLOCK_SIZE);

    srand(time(NULL));
    int seed = rand();

    hiprandState *devStates;
    CHECK(hipMalloc((void **)&devStates, blocksperdict*BLOCK_SIZE*sizeof(hiprandState)));

    normfill<<<blocksperdict,BLOCK_SIZE>>>(D, n*m, devStates, seed);
    CHECK(hipDeviceSynchronize());

    double *tmpcol,*partialNormBlocks,norm;
    int blockspercol = ceil(n*1.0/BLOCK_SIZE);
    CHECK(hipMalloc(&tmpcol, blockspercol*BLOCK_SIZE*sizeof(double)));
    CHECK(hipMemset(tmpcol, 0, blockspercol*BLOCK_SIZE*sizeof(double)));
    CHECK(hipHostMalloc(&partialNormBlocks, blockspercol*sizeof(double)));

    for(int i=0; i<m; i++){

        CHECK(hipMemcpy(tmpcol, &D[i*n], n*sizeof(double), hipMemcpyDeviceToDevice));

        //CALCOLA NORMA CON vector2norm
        vector2norm<<<blockspercol,BLOCK_SIZE>>>(tmpcol);
        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy(partialNormBlocks, tmpcol, blockspercol*sizeof(double), hipMemcpyDeviceToHost));
        norm = 0;
        for(int j=0; j<blockspercol; j++)
            norm += partialNormBlocks[j];
        norm = sqrt(norm);

        //CHIAMA KERNEL CHE DIVIDE OGNI ELEMENTO PER LA NORMA
        divide<<<blockspercol,BLOCK_SIZE>>>(&D[i*n], norm, n);
        CHECK(hipDeviceSynchronize());
    }
}

void generateAlpha(double *alpha, int m, int k){

    int blocksperk = ceil(k*1.0/BLOCK_SIZE);

    srand(time(NULL));
    int seed = rand();

    hiprandState *devStates;
    CHECK(hipMalloc(&devStates, blocksperk*BLOCK_SIZE*sizeof(hiprandState)));

    double *d_kvalues,h_kvalues[k];
    CHECK(hipMalloc(&d_kvalues, k*sizeof(double)));

    normfill<<<blocksperk,BLOCK_SIZE>>>(d_kvalues, k, devStates, seed);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(h_kvalues, d_kvalues, k*sizeof(double), hipMemcpyDeviceToHost));

    double h_alpha[m];
    memset(h_alpha, 0, m*sizeof(double));
    for(int i=0; i<k; i++){
        int idx = rand()%m;
        if(h_alpha[idx] != 0)
            i--;
        else
            h_alpha[idx] = h_kvalues[i];
    }

    CHECK(hipMemcpy(alpha, h_alpha, m * sizeof(double), hipMemcpyHostToDevice));

}
