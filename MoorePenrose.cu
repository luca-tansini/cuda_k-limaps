#include "hip/hip_runtime.h"
#include <float.h>
#include <hipsolver.h>

/*
Kernel that takes the vector of singular values S and produces the pseudo inverse of the diagonal matrix of S. According to the instances, we assume n >= m. Since [n x m] would be the dimension of S diag matrix, and we also have to transpose the diag matrix, its pseudo inverse will be [m x n], with leading dimension m.
The elements on the main diagonal are to be inverted only if non-zero. To determine what is zero we use a threshold based on the machine FLT_EPSILON.
*/
__global__ void calculateDiagPseudoInv(float *S, float *SPseudoInv, int n, int m){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n)
        if(fabsf(S[idx]) > FLT_EPSILON)
            SPseudoInv[idx*m + idx] = 1/S[idx];
        else
            SPseudoInv[idx*m + idx] = S[idx];
}

/*
This function calcuates the Moore-Penrose inverse matrix of the input matrix A (n*m, with n > m), leaving the result in APseudoInv, assumed preallocated.

The pseudoinverse is computed via SVD.
If SVD(A) = U*S*V^T --> A^+ = V * S^+ * U^T, where S^+ is obtained replacing each non-zero element on the diagonal with its reciprocal.

The cuSOLVER libraries used to calculate the SVD need the input matrix to be n x m with n >= m.
*/
void MoorePenroseInverse(float *A, int n, int m, float *APseudoInv){

    if(n < m){
        printf("error: n must be >= m! (you can transpose the input matrix and then transpose the result to work with matrices that have less rows than columns)\n");
        return;
    }

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //transpose matrix A into APseudoInv
    float alpha=1,beta=0;
    //CHECK_CUBLAS(hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &alpha, A, m, &beta, A, n, APseudoInv, n));

    //Get dimension needed for the workspace buffer and allocate it
    int bufferDim;
    float *buffer;
    CHECK_CUSOLVER(hipsolverDnSgesvd_bufferSize(cusolverHandle, n, m, &bufferDim));
    CHECK(hipMalloc(&buffer,bufferDim));

    //Allocate U,S,V_T
    float *U,*S,*V_T;
    CHECK(hipMalloc(&U, n*n*sizeof(float)));
    CHECK(hipMalloc(&S1, m*sizeof(float)));
    CHECK(hipMalloc(&V_T1, m*m*sizeof(float)));

    //Calculate SVD with cuSOLVER
    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnSgesvd(cusolverHandle, 'A', 'A', n, m, APseudoInv, n, S, U, n, V_T, m, buffer, bufferDim, NULL, dev_info));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //Calculate S^+
    float *SPseudoInv;
    CHECK(hipMalloc(&SPseudoInv, m*n*sizeof(float)));
    CHECK(hipMemset(SPseudoInv, 0, m*n*sizeof(float)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(m*1.0/BLOCK_SIZE),1,1);
    calculateDiagPseudoInv<<<dimGrid,dimBlock>>>(S1, S1PseudoInv, n, m);
    CHECK(hipDeviceSynchronize());

    //DEBUG
    float *h_SPseudoInv;
    CHECK(hipHostMalloc(&h_SPseudoInv, m*n*sizeof(float)));
    CHECK(hipMemcpy(h_SPseudoInv, SPseudoInv, m*n*sizeof(float), hipMemcpyDeviceToHost));
    printColumnMajorMatrix(h_SPseudoInv, m, n);
    CHECK(hipHostFree(h_SPseudoInv));
    //END DEBUG

    //calculate APseudoInv = V_T^T * S^+ * U^T
    //APseudoInv = V_T^T * S1^+
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, m, &alpha, V_T, m, SPseudoInv, m, &beta, APseudoInv, m));
    //APseudoInv *= U^T
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, n, &alpha, APseudoInv, m, U, n, &beta, APseudoInv, m));

    //Free memory
    CHECK(hipFree(buffer));
    CHECK(hipFree(U1));
    CHECK(hipFree(S1));
    CHECK(hipFree(S1PseudoInv));
    CHECK(hipFree(V_T1));

}
