#include "hip/hip_runtime.h"
#include <float.h>
#include <hipsolver.h>

#ifndef _COMMON_H
    #include "common.h"
#endif

#ifndef _MATRIX_PRINT_H
    #include "matrixPrint.h"
#endif

#define BLOCK_SIZE 256

/*
Kernel that takes the vector of singular values S (of length m) and produces the pseudo inverse of the diagonal matrix of S. According to the instances, we assume n >= m. Since [n x m] would be the dimension of S diag matrix, and we also have to transpose the diag matrix, its pseudo inverse will be [m x n], with leading dimension m.
The elements on the main diagonal are to be inverted only if non-zero. To determine what is zero we use a threshold based on the machine FLT_EPSILON.
*/
__global__ void calculateDiagPseudoInv(float *S, float *SPseudoInv, int n, int m){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < m){
        if(fabsf(S[idx]) > FLT_EPSILON)
            SPseudoInv[idx*m + idx] = 1/S[idx];
        else
            SPseudoInv[idx*m + idx] = S[idx];
    }
}

/*
This function calcuates the Moore-Penrose inverse matrix of the input matrix A (n*m, with n > m), leaving the result in APseudoInv, assumed preallocated.

The pseudoinverse is computed via SVD.
If SVD(A) = U*S*V^T --> A^+ = V * S^+ * U^T, where S^+ is obtained replacing each non-zero element on the diagonal with its reciprocal and transposing.

The cuSOLVER libraries used to calculate the SVD need the input matrix to be n x m with n >= m.
*/
void MoorePenroseInverse(float *A, int n, int m, float *APseudoInv){

    if(n < m){
        printf("error: n must be >= m! (you can transpose the input matrix and then transpose the result to work with matrices that have less rows than columns)\n");
        return;
    }

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Get dimension needed for the workspace buffer and allocate it
    int bufferDim;
    float *buffer;
    CHECK_CUSOLVER(hipsolverDnSgesvd_bufferSize(cusolverHandle, n, m, &bufferDim));
    CHECK(hipMalloc(&buffer,bufferDim*sizeof(float)));

    //Allocate U,S,V_T
    float *U,*S,*V_T;
    CHECK(hipMalloc(&U, n*n*sizeof(float)));
    CHECK(hipMalloc(&S, m*sizeof(float)));
    CHECK(hipMalloc(&V_T, m*m*sizeof(float)));

    //Calculate SVD with cuSOLVER

    float *Acopy; //we use a copy of A because apparently gesvd destroys input matrix
    CHECK(hipMalloc(&Acopy, n*m*sizeof(float)));
    CHECK(hipMemcpy(Acopy, A, n*m*sizeof(float), hipMemcpyDeviceToDevice));

    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnSgesvd(cusolverHandle, 'A', 'A', n, m, Acopy, n, S, U, n, V_T, m, buffer, bufferDim, NULL, dev_info));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //Calculate S^+
    float *SPseudoInv;
    CHECK(hipMalloc(&SPseudoInv, m*n*sizeof(float)));
    CHECK(hipMemset(SPseudoInv, 0, m*n*sizeof(float)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(m*1.0/BLOCK_SIZE),1,1);
    calculateDiagPseudoInv<<<dimGrid,dimBlock>>>(S, SPseudoInv, n, m);
    CHECK(hipDeviceSynchronize());

    //calculate APseudoInv = V_T^T * S^+ * U^T
    //APseudoInv = V_T^T * S^+
    float alpha=1,beta=0;
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, m, &alpha, V_T, m, SPseudoInv, m, &beta, APseudoInv, m));
    //APseudoInv *= U^T
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, n, &alpha, APseudoInv, m, U, n, &beta, APseudoInv, m));

    //Free memory
    CHECK(hipFree(buffer));
    CHECK(hipFree(U));
    CHECK(hipFree(S));
    CHECK(hipFree(SPseudoInv));
    CHECK(hipFree(V_T));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandle));

}

void TransposedMoorePenroseInverse(float *A, int n, int m, float *APseudoInv){

    float *AT,*APseudoInvT;
    CHECK(hipMalloc(&AT, m*n*sizeof(float)));
    CHECK(hipMalloc(&APseudoInvT, n*m*sizeof(float)));

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    float alpha = 1, beta = 0;

    //Transpose A
    CHECK_CUBLAS(hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, A, n, &beta, A, n, AT, m));

    //Call MoorePenroseInverse
    MoorePenroseInverse(AT, m, n, APseudoInvT);

    //Transpose APseudoInvT
    CHECK_CUBLAS(hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, APseudoInvT, n, &beta, APseudoInvT, n, APseudoInv, m));

    CHECK(hipFree(AT));
    CHECK(hipFree(APseudoInvT));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

}

void DebugMoorePenroseInverse(float *A, int n, int m, float *APseudoInv){

    if(n < m){
        printf("error: n must be >= m! (you can transpose the input matrix and then transpose the result to work with matrices that have less rows than columns)\n");
        return;
    }

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Get dimension needed for the workspace buffer and allocate it
    int bufferDim;
    float *buffer;
    CHECK_CUSOLVER(hipsolverDnSgesvd_bufferSize(cusolverHandle, n, m, &bufferDim));
    CHECK(hipMalloc(&buffer,bufferDim*sizeof(float)));

    //Allocate U,S,V_T
    float *U,*S,*V_T;
    CHECK(hipMalloc(&U, n*n*sizeof(float)));
    CHECK(hipMalloc(&S, m*sizeof(float)));
    CHECK(hipMalloc(&V_T, m*m*sizeof(float)));

    //Calculate SVD with cuSOLVER

    float *Acopy; //we use a copy of A because apparently gesvd destroys input matrix
    CHECK(hipMalloc(&Acopy, n*m*sizeof(float)));
    CHECK(hipMemcpy(Acopy, A, n*m*sizeof(float), hipMemcpyDeviceToDevice));

    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnSgesvd(cusolverHandle, 'A', 'A', n, m, Acopy, n, S, U, n, V_T, m, buffer, bufferDim, NULL, dev_info));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //Calculate S^+
    float *SPseudoInv;
    CHECK(hipMalloc(&SPseudoInv, m*n*sizeof(float)));
    CHECK(hipMemset(SPseudoInv, 0, m*n*sizeof(float)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(m*1.0/BLOCK_SIZE),1,1);
    calculateDiagPseudoInv<<<dimGrid,dimBlock>>>(S, SPseudoInv, n, m);
    CHECK(hipDeviceSynchronize());

    //*********************************DEBUG************************************

    float *h_U,*h_S,*h_V_T,*h_SPseudoInv;
    CHECK(hipHostMalloc(&h_U, n*n*sizeof(float)));
    CHECK(hipHostMalloc(&h_S, m*sizeof(float)));
    CHECK(hipHostMalloc(&h_V_T, m*m*sizeof(float)));
    CHECK(hipHostMalloc(&h_SPseudoInv, m*n*sizeof(float)));
    CHECK(hipMemcpy(h_U, U, n*n*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_S, S, m*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_V_T, V_T, m*m*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_SPseudoInv, SPseudoInv, m*n*sizeof(float), hipMemcpyDeviceToHost));

    printf("\nU:\n");
    printColumnMajorMatrixForPython(h_U, n, n);

    printf("\nS:\n");
    printColumnMajorMatrixForPython(h_S, 1, m);

    printf("\nVT:\n");
    printColumnMajorMatrixForPython(h_V_T, m, m);

    printf("\nSPseudoInv:\n");
    printColumnMajorMatrixForPython(h_SPseudoInv, m, n);
    printf("\n");

    //******************************END DEBUG***********************************

    //calculate APseudoInv = V_T^T * S^+ * U^T
    //APseudoInv = V_T^T * S^+
    float alpha=1,beta=0;
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, m, &alpha, V_T, m, SPseudoInv, m, &beta, APseudoInv, m));
    //APseudoInv *= U^T
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, n, &alpha, APseudoInv, m, U, n, &beta, APseudoInv, m));

    //Free memory
    CHECK(hipFree(buffer));
    CHECK(hipFree(U));
    CHECK(hipFree(S));
    CHECK(hipFree(SPseudoInv));
    CHECK(hipFree(V_T));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandle));

}

void TransposeDebugMoorePenroseInverse(float *A, int n, int m, float *APseudoInv){

    float *AT,*APseudoInvT;
    CHECK(hipMalloc(&AT, m*n*sizeof(float)));
    CHECK(hipMalloc(&APseudoInvT, n*m*sizeof(float)));

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    float alpha = 1, beta = 0;

    //Transpose A
    CHECK_CUBLAS(hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, A, n, &beta, A, n, AT, m));

    //DEBUG
    float *h_AT;
    CHECK(hipHostMalloc(&h_AT, m*n*sizeof(float)));
    CHECK(hipMemcpy(h_AT, AT, m*n*sizeof(float), hipMemcpyDeviceToHost));
    printf("A^T:\n");
    printColumnMajorMatrixForPython(h_AT,m,n);
    printf("\n");
    //END DEBUG

    //Call MoorePenroseInverse
    DebugMoorePenroseInverse(AT, m, n, APseudoInvT);

    //Transpose APseudoInvT
    CHECK_CUBLAS(hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, APseudoInvT, n, &beta, APseudoInvT, n, APseudoInv, m));

    CHECK(hipFree(AT));
    CHECK(hipFree(APseudoInvT));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

}

//The performed check is A * Apinv * A =?= A
int CheckPseudoInverse(float *A, int n, int m, float *Apinv){

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //tmp = A * Apinv
    float *tmp,alpha=1,beta=0;
    CHECK(hipMalloc(&tmp, n*n*sizeof(float)));

    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, m, &alpha, A, n, Apinv, m, &beta, tmp, n));

    //tmp2 = tmp * A
    float *tmp2;
    CHECK(hipMalloc(&tmp2, n*m*sizeof(float)));

    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, n, &alpha, tmp, n, A, n, &beta, tmp2, n));

    //tmp2 =?= A
    float *h_tmp2,*h_A;
    CHECK(hipHostMalloc(&h_tmp2, n*m*sizeof(float)));
    CHECK(hipMemcpy(h_tmp2, tmp2, n*m*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipHostMalloc(&h_A, n*m*sizeof(float)));
    CHECK(hipMemcpy(h_A, A, n*m*sizeof(float), hipMemcpyDeviceToHost));

    int i;
    for(i=0; i<n*m; i++)
        if(fabs(h_A[i] - h_tmp2[i]) > 1e-3){
            printf("at index %d diff is: %f\n",i, h_A[i] - h_tmp2[i]);
            break;
        }

    int ret=1;
    if(i < n*m)
        ret = 0;

    CHECK(hipFree(tmp));
    CHECK(hipFree(tmp2));
    CHECK(hipHostFree(h_tmp2));
    CHECK(hipHostFree(h_A));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

    return ret;
}
