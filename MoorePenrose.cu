#include "hip/hip_runtime.h"
#include <float.h>
#include <hipsolver.h>

#ifndef _COMMON_H
    #include "common.h"
#endif

#ifndef _MATRIX_PRINT_H
    #include "matrixPrint.h"
#endif

#define BLOCK_SIZE 256

/*
Kernel che prende il vettore di valori singolari S (di lunghezza m) e produce la pseudo inversa della matrice diagonale di S. Poichè [n x m] sarebbe la dimensione della matrice diagonale di S, e nel calcolo della pseudoinversa dobbiamo anche trasporre la matrice, la pseudoinversa avrà dimensione [m x n], con leading dimension m.
Gli elementi della diagonale devono essere invertiti solo se diversi da 0. Per determinare che cosa è zero, usiamo la costante DBL_EPSILON della macchina.
*/
__global__ void calculateDiagPinv(double *S, double *Spinv, int n, int m){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < m && idx < n){
        if(fabsf(S[idx]) > DBL_EPSILON)
            Spinv[idx*m + idx] = 1/S[idx];
        else
            Spinv[idx*m + idx] = 0;
    }
}

/*
Questa funzione calcola la pseudoinversa di Moore-Penrose della matrice in input A (n*m, con n > m), restituendo il risultato in Apinv (che si assume sia preallocato).

La pseudoinversa è calcolata tramite decomposizione SVD.
Se SVD(A) = U*S*V^T --> A^+ = V * S^+ * U^T, dove S^+ è ottenuta sostituendo ogni elemento non-zero della diagonale con il suo reciproco e trasponendo.

Le librerie cuSOLVER usate per calcolare l'SVD hanno bisogno che la matrice in input sia n x m con n >= m.
*/
void MoorePenroseInverse(double *A, int n, int m, double *Apinv){

    if(n < m){
        printf("error: n must be >= m! (you can transpose the input matrix and then transpose the result to work with matrices that have less rows than columns)\n");
        return;
    }

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Calcola la dimensione per il buffer  di lavoro e lo alloca
    int bufferDim;
    double *buffer;
    CHECK_CUSOLVER(hipsolverDnDgesvd_bufferSize(cusolverHandle, n, m, &bufferDim));
    CHECK(hipMalloc(&buffer,bufferDim*sizeof(double)));

    //Alloca U,S,VT
    double *U,*S,*VT;
    CHECK(hipMalloc(&U, n*n*sizeof(double)));
    CHECK(hipMalloc(&S, m*sizeof(double)));
    CHECK(hipMalloc(&VT, m*m*sizeof(double)));

    //Calcola SVD con cuSOLVER
    double *Acopy; //usiamo una copia di A perchè gesvd distrugge la matrice in input
    CHECK(hipMalloc(&Acopy, n*m*sizeof(double)));
    CHECK(hipMemcpy(Acopy, A, n*m*sizeof(double), hipMemcpyDeviceToDevice));

    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnDgesvd(cusolverHandle, 'A', 'A', n, m, Acopy, n, S, U, n, VT, m, buffer, bufferDim, NULL, dev_info));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //Calcola S^+
    double *Spinv;
    CHECK(hipMalloc(&Spinv, m*n*sizeof(double)));
    CHECK(hipMemset(Spinv, 0, m*n*sizeof(double)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(m*1.0/BLOCK_SIZE),1,1);
    calculateDiagPinv<<<dimGrid,dimBlock>>>(S, Spinv, n, m);
    CHECK(hipDeviceSynchronize());

    //Calcola Apinv = VT^T * S^+ * U^T
    double alpha=1,beta=0,*tmp;
    CHECK(hipMalloc(&tmp, m*n*sizeof(double)));

    //tmp = VT^T * S^+
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, m, &alpha, VT, m, Spinv, m, &beta, tmp, m));

    //Apinv = tmp * U^T
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, n, &alpha, tmp, m, U, n, &beta, Apinv, m));

    //Free della memoria
    CHECK(hipFree(buffer));
    CHECK(hipFree(U));
    CHECK(hipFree(S));
    CHECK(hipFree(Spinv));
    CHECK(hipFree(VT));
    CHECK(hipFree(tmp));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandle));

}

/*
Questa versione usa hipsolverDnSgesvdj che usa il metodo di Jacobi per la SVD decomposition.
Non ci sono vincoli su n e m.
hipsolverDnSgesvdj restituisce V invece che VH
*/
void JacobiMoorePenroseInverse(double *A, int n, int m, double *Apinv){

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Alloca U,S,V
    double *U,*S,*V;
    CHECK(hipMalloc(&U, n*n*sizeof(double)));
    CHECK(hipMalloc(&S, m*sizeof(double)));
    CHECK(hipMalloc(&V, m*m*sizeof(double)));

    //Calcola SVD con cuSOLVER
    double *Acopy; //usiamo una copia di A perchè gesvdj distrugge la matrice in input
    CHECK(hipMalloc(&Acopy, n*m*sizeof(double)));
    CHECK(hipMemcpy(Acopy, A, n*m*sizeof(double), hipMemcpyDeviceToDevice));

    //Set up dei parametri di hipsolverDnDgesvdj
    int bufferDim;
    double *buffer;
    hipsolverGesvdjInfo_t gesvdj_params = NULL; //parametri di default
    hipsolverDnCreateGesvdjInfo(&gesvdj_params);
    CHECK_CUSOLVER(hipsolverDnDgesvdj_bufferSize(cusolverHandle, HIPSOLVER_EIG_MODE_VECTOR, 0, n, m, Acopy, n, S, U, n, V, m, &bufferDim, gesvdj_params));
    CHECK(hipMalloc(&buffer,bufferDim*sizeof(double)));

    //Chiama hipsolverDnDgesvdj
    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnDgesvdj(cusolverHandle, HIPSOLVER_EIG_MODE_VECTOR, 0, n, m, Acopy, n, S, U, n, V, m, buffer, bufferDim, dev_info, gesvdj_params));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //Calcola S^+
    double *Spinv;
    CHECK(hipMalloc(&Spinv, m*n*sizeof(double)));
    CHECK(hipMemset(Spinv, 0, m*n*sizeof(double)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(m*1.0/BLOCK_SIZE),1,1);
    calculateDiagPinv<<<dimGrid,dimBlock>>>(S, Spinv, n, m);
    CHECK(hipDeviceSynchronize());

    //Calcola Apinv = V * S^+ * U^T
    double alpha=1,beta=0,*tmp;
    CHECK(hipMalloc(&tmp, m*n*sizeof(double)));

    //tmp = V * S^+
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, m, &alpha, V, m, Spinv, m, &beta, tmp, m));

    //Apinv = tmp * U^T
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, n, &alpha, tmp, m, U, n, &beta, Apinv, m));

    //Free della memoria
    CHECK(hipFree(buffer));
    CHECK(hipFree(U));
    CHECK(hipFree(S));
    CHECK(hipFree(Spinv));
    CHECK(hipFree(V));
    CHECK(hipFree(tmp));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandle));

}

/*
Funzione che verifica la correttezza del calcolo della pseudoinversa.
Il controllo effettuato si basa sulle proprietà della pseudoinversa ed è: A * Apinv * A =?= A
*/
int CheckPseudoinverse(double *A, int n, int m, double *Apinv){

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //tmp = A * Apinv
    double *tmp,alpha=1,beta=0;
    CHECK(hipMalloc(&tmp, n*n*sizeof(double)));

    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, m, &alpha, A, n, Apinv, m, &beta, tmp, n));

    //tmp2 = tmp * A
    double *tmp2;
    CHECK(hipMalloc(&tmp2, n*m*sizeof(double)));

    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, n, &alpha, tmp, n, A, n, &beta, tmp2, n));

    //tmp2 =?= A
    double *h_tmp2,*h_A;
    CHECK(hipHostMalloc(&h_tmp2, n*m*sizeof(double)));
    CHECK(hipMemcpy(h_tmp2, tmp2, n*m*sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipHostMalloc(&h_A, n*m*sizeof(double)));
    CHECK(hipMemcpy(h_A, A, n*m*sizeof(double), hipMemcpyDeviceToHost));

    int i;
    for(i=0; i<n*m; i++)
        if(fabs(h_A[i] - h_tmp2[i]) > 1e-5){
            printf("at index %d diff is: %f\n",i, h_A[i] - h_tmp2[i]);
            break;
        }

    int ret=1;
    if(i < n*m)
        ret = 0;

    CHECK(hipFree(tmp));
    CHECK(hipFree(tmp2));
    CHECK(hipHostFree(h_tmp2));
    CHECK(hipHostFree(h_A));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

    return ret;
}
