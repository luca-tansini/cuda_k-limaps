#include "hip/hip_runtime.h"
#include <float.h>
#include <hipsolver.h>

#ifndef _COMMON_H
    #include "common.h"
#endif

#ifndef _MATRIX_PRINT_H
    #include "matrixPrint.h"
#endif

#define BLOCK_SIZE 256

/*
Kernel that takes the vector of singular values S (of length m) and produces the pseudo inverse of the diagonal matrix of S. According to the instances, we assume n >= m. Since [n x m] would be the dimension of S diag matrix, and we also have to transpose the diag matrix, its pseudo inverse will be [m x n], with leading dimension m.
The elements on the main diagonal are to be inverted only if non-zero. To determine what is zero we use a threshold based on the machine FLT_EPSILON.
*/
__global__ void calculateDiagPseudoInv(float *S, float *SPseudoInv, int n, int m){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < m){
        if(fabsf(S[idx]) > FLT_EPSILON)
            SPseudoInv[idx*m + idx] = 1/S[idx];
        else
            SPseudoInv[idx*m + idx] = S[idx];
    }
}

/*
This function calcuates the Moore-Penrose inverse matrix of the input matrix A (n*m, with n > m), leaving the result in APseudoInv, assumed preallocated.

The pseudoinverse is computed via SVD.
If SVD(A) = U*S*V^T --> A^+ = V * S^+ * U^T, where S^+ is obtained replacing each non-zero element on the diagonal with its reciprocal and transposing.

The cuSOLVER libraries used to calculate the SVD need the input matrix to be n x m with n >= m.
*/
void MoorePenroseInverse(float *A, int n, int m, float *APseudoInv){

    if(n < m){
        printf("error: n must be >= m! (you can transpose the input matrix and then transpose the result to work with matrices that have less rows than columns)\n");
        return;
    }

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Get dimension needed for the workspace buffer and allocate it
    int bufferDim;
    float *buffer;
    CHECK_CUSOLVER(hipsolverDnSgesvd_bufferSize(cusolverHandle, n, m, &bufferDim));
    CHECK(hipMalloc(&buffer,bufferDim*sizeof(float)));

    //Allocate U,S,V_T
    float *U,*S,*V_T;
    CHECK(hipMalloc(&U, n*n*sizeof(float)));
    CHECK(hipMalloc(&S, m*sizeof(float)));
    CHECK(hipMalloc(&V_T, m*m*sizeof(float)));

    //Calculate SVD with cuSOLVER

    float *Acopy; //we use a copy of A because apparently gesvd destroys input matrix
    CHECK(hipMalloc(&Acopy, n*m*sizeof(float)));
    CHECK(hipMemcpy(Acopy, A, n*m*sizeof(float), hipMemcpyDeviceToDevice));

    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnSgesvd(cusolverHandle, 'A', 'A', n, m, Acopy, n, S, U, n, V_T, m, buffer, bufferDim, NULL, dev_info));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //Calculate S^+
    float *SPseudoInv;
    CHECK(hipMalloc(&SPseudoInv, m*n*sizeof(float)));
    CHECK(hipMemset(SPseudoInv, 0, m*n*sizeof(float)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(m*1.0/BLOCK_SIZE),1,1);
    calculateDiagPseudoInv<<<dimGrid,dimBlock>>>(S, SPseudoInv, n, m);
    CHECK(hipDeviceSynchronize());

    //calculate APseudoInv = V_T^T * S^+ * U^T
    //APseudoInv = V_T^T * S^+
    float alpha=1,beta=0;
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, m, &alpha, V_T, m, SPseudoInv, m, &beta, APseudoInv, m));
    //APseudoInv *= U^T
    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, n, &alpha, APseudoInv, m, U, n, &beta, APseudoInv, m));

    //Free memory
    CHECK(hipFree(buffer));
    CHECK(hipFree(U));
    CHECK(hipFree(S));
    CHECK(hipFree(SPseudoInv));
    CHECK(hipFree(V_T));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandle));

}

void TransposedMoorePenroseInverse(float *A, int n, int m, float *APseudoInv){

    float *AT,*APseudoInvT;
    CHECK(hipMalloc(&AT, m*n*sizeof(float)));
    CHECK(hipMalloc(&APseudoInvT, n*m*sizeof(float)));

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    float alpha = 1, beta = 0;

    //Transpose A
    CHECK_CUBLAS(hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, A, n, &beta, A, n, AT, m));

    //Call MoorePenroseInverse
    MoorePenroseInverse(AT, m, n, APseudoInvT);

    //Transpose APseudoInvT
    CHECK_CUBLAS(hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, APseudoInvT, n, &beta, APseudoInvT, n, APseudoInv, m));

    CHECK(hipFree(AT));
    CHECK(hipFree(APseudoInvT));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

}

int CheckPseudoInverse(float *A, int n, int m, float *Apinv){

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    float *id,*d_id,alpha=1,beta=0;
    CHECK(hipMalloc(&d_id, m*m*sizeof(float)));

    CHECK_CUBLAS(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, m, n, &alpha, Apinv, m, A, n, &beta, d_id, m));

    CHECK(hipHostMalloc(&id, m*m*sizeof(float)));
    CHECK(hipMemcpy(id, d_id, m*m*sizeof(float), hipMemcpyDeviceToHost));

    int i;
    for(i=0;i<m;i++)
        for(int j=0; j<m; j++){
            if(i == j && abs(1-id[j*m+i]) > 1e-4) break;
            if(i != j && abs(0-id[j*m+i]) > 1e-4) break;
        }

    int ret;
    if(i < m){
        printf("Apinv * A:\n");
        printColumnMajorMatrix(id, m, m);
        ret = 0;
    }
    else{
        ret = 1;
    }

    CHECK(hipFree(d_id));
    CHECK(hipHostFree(id));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    return ret;
}
