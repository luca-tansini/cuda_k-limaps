#include "hip/hip_runtime.h"
#include <float.h>
#include <hipsolver.h>
#include "singular_value_decomposition.h"

#ifndef _COMMON_H
    #include "common.h"
#endif

#ifndef _MATRIX_PRINT_H
    #include "matrixPrint.h"
#endif

#define BLOCK_SIZE 256

/*
Kernel that takes the vector of singular values S (of length m) and produces the pseudo inverse of the diagonal matrix of S. Since [n x m] would be the dimension of S diag matrix, and we also have to transpose the diag matrix, its pseudo inverse will be [m x n], with leading dimension m.
The elements on the main diagonal are to be inverted only if non-zero. To determine what is zero we use a threshold based on the machine DBL_EPSILON.
*/
__global__ void calculateDiagPinv(double *S, double *Spinv, int n, int m){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < m && idx < n){
        if(fabsf(S[idx]) > DBL_EPSILON)
            Spinv[idx*m + idx] = 1/S[idx];
        else
            Spinv[idx*m + idx] = 0;
    }
}

/*
This version uses hipsolverDnSgesvdj which uses the Jacobi method for SVD decomposition.
There is no constraint on n and m.
hipsolverDnSgesvdj returns V instead of VH
*/
void JacobiMoorePenroseInverse(double *A, int n, int m, double *Apinv){

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Allocate U,S,V
    double *U,*S,*V;
    CHECK(hipMalloc(&U, n*n*sizeof(double)));
    CHECK(hipMalloc(&S, m*sizeof(double)));
    CHECK(hipMalloc(&V, m*m*sizeof(double)));

    //Calculate SVD with cuSOLVER
    double *Acopy; //we use a copy of A because gesvdj destroys input matrix
    CHECK(hipMalloc(&Acopy, n*m*sizeof(double)));
    CHECK(hipMemcpy(Acopy, A, n*m*sizeof(double), hipMemcpyDeviceToDevice));

    //Set up hipsolverDnDgesvdj parameters
    int bufferDim;
    double *buffer;
    hipsolverGesvdjInfo_t gesvdj_params = NULL; //deafult params
    hipsolverDnCreateGesvdjInfo(&gesvdj_params);
    CHECK_CUSOLVER(hipsolverDnDgesvdj_bufferSize(cusolverHandle, HIPSOLVER_EIG_MODE_VECTOR, 0, n, m, Acopy, n, S, U, n, V, m, &bufferDim, gesvdj_params));
    CHECK(hipMalloc(&buffer,bufferDim*sizeof(double)));

    //Call hipsolverDnDgesvdj
    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnDgesvdj(cusolverHandle, HIPSOLVER_EIG_MODE_VECTOR, 0, n, m, Acopy, n, S, U, n, V, m, buffer, bufferDim, dev_info, gesvdj_params));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //Calculate S^+
    double *Spinv;
    CHECK(hipMalloc(&Spinv, m*n*sizeof(double)));
    CHECK(hipMemset(Spinv, 0, m*n*sizeof(double)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(m*1.0/BLOCK_SIZE),1,1);
    calculateDiagPinv<<<dimGrid,dimBlock>>>(S, Spinv, n, m);
    CHECK(hipDeviceSynchronize());

    //calculate Apinv = V * S^+ * U^T
    double alpha=1,beta=0,*tmp;
    CHECK(hipMalloc(&tmp, m*n*sizeof(double)));

    //tmp = V * S^+
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, m, &alpha, V, m, Spinv, m, &beta, tmp, m));

    //Apinv = tmp * U^T
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, n, &alpha, tmp, m, U, n, &beta, Apinv, m));

    //Free memory
    CHECK(hipFree(buffer));
    CHECK(hipFree(U));
    CHECK(hipFree(S));
    CHECK(hipFree(Spinv));
    CHECK(hipFree(V));
    CHECK(hipFree(tmp));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandle));

}

/*
This function calcuates the Moore-Penrose inverse matrix of the input matrix A (n*m, with n > m), leaving the result in Apinv, assumed preallocated.

The pseudoinverse is computed via SVD.
If SVD(A) = U*S*V^T --> A^+ = V * S^+ * U^T, where S^+ is obtained replacing each non-zero element on the diagonal with its reciprocal and transposing.

The cuSOLVER libraries used to calculate the SVD need the input matrix to be n x m with n >= m.
*/
void MoorePenroseInverse(double *A, int n, int m, double *Apinv){

    if(n < m){
        printf("error: n must be >= m! (you can transpose the input matrix and then transpose the result to work with matrices that have less rows than columns)\n");
        return;
    }

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Get dimension needed for the workspace buffer and allocate it
    int bufferDim;
    double *buffer;
    CHECK_CUSOLVER(hipsolverDnDgesvd_bufferSize(cusolverHandle, n, m, &bufferDim));
    CHECK(hipMalloc(&buffer,bufferDim*sizeof(double)));

    //Allocate U,S,VT
    double *U,*S,*VT;
    CHECK(hipMalloc(&U, n*n*sizeof(double)));
    CHECK(hipMalloc(&S, m*sizeof(double)));
    CHECK(hipMalloc(&VT, m*m*sizeof(double)));

    //Calculate SVD with cuSOLVER
    double *Acopy; //we use a copy of A because apparently gesvd destroys input matrix
    CHECK(hipMalloc(&Acopy, n*m*sizeof(double)));
    CHECK(hipMemcpy(Acopy, A, n*m*sizeof(double), hipMemcpyDeviceToDevice));

    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnDgesvd(cusolverHandle, 'A', 'A', n, m, Acopy, n, S, U, n, VT, m, buffer, bufferDim, NULL, dev_info));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //Calculate S^+
    double *Spinv;
    CHECK(hipMalloc(&Spinv, m*n*sizeof(double)));
    CHECK(hipMemset(Spinv, 0, m*n*sizeof(double)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(m*1.0/BLOCK_SIZE),1,1);
    calculateDiagPinv<<<dimGrid,dimBlock>>>(S, Spinv, n, m);
    CHECK(hipDeviceSynchronize());

    //calculate Apinv = VT^T * S^+ * U^T
    double alpha=1,beta=0,*tmp;
    CHECK(hipMalloc(&tmp, m*n*sizeof(double)));

    //tmp = VT^T * S^+
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, m, &alpha, VT, m, Spinv, m, &beta, tmp, m));

    //Apinv = tmp * U^T
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, n, &alpha, tmp, m, U, n, &beta, Apinv, m));

    //Free memory
    CHECK(hipFree(buffer));
    CHECK(hipFree(U));
    CHECK(hipFree(S));
    CHECK(hipFree(Spinv));
    CHECK(hipFree(VT));
    CHECK(hipFree(tmp));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandle));

}

void TransposedMoorePenroseInverse(double *A, int n, int m, double *Apinv){

    double *AT,*ApinvT;
    CHECK(hipMalloc(&AT, m*n*sizeof(double)));
    CHECK(hipMalloc(&ApinvT, n*m*sizeof(double)));

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    double alpha = 1, beta = 0;

    //Transpose A
    CHECK_CUBLAS(hipblasDgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, A, n, &beta, A, n, AT, m));

    //Call MoorePenroseInverse
    MoorePenroseInverse(AT, m, n, ApinvT);

    //Transpose ApinvT
    CHECK_CUBLAS(hipblasDgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, ApinvT, n, &beta, ApinvT, n, Apinv, m));

    CHECK(hipFree(AT));
    CHECK(hipFree(ApinvT));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

}

//The performed check is A * Apinv * A =?= A
int CheckPseudoinverse(double *A, int n, int m, double *Apinv){

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //tmp = A * Apinv
    double *tmp,alpha=1,beta=0;
    CHECK(hipMalloc(&tmp, n*n*sizeof(double)));

    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, m, &alpha, A, n, Apinv, m, &beta, tmp, n));

    //tmp2 = tmp * A
    double *tmp2;
    CHECK(hipMalloc(&tmp2, n*m*sizeof(double)));

    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, n, &alpha, tmp, n, A, n, &beta, tmp2, n));

    //tmp2 =?= A
    double *h_tmp2,*h_A;
    CHECK(hipHostMalloc(&h_tmp2, n*m*sizeof(double)));
    CHECK(hipMemcpy(h_tmp2, tmp2, n*m*sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipHostMalloc(&h_A, n*m*sizeof(double)));
    CHECK(hipMemcpy(h_A, A, n*m*sizeof(double), hipMemcpyDeviceToHost));

    int i;
    for(i=0; i<n*m; i++)
        if(fabs(h_A[i] - h_tmp2[i]) > 1e-5){
            printf("at index %d diff is: %f\n",i, h_A[i] - h_tmp2[i]);
            break;
        }

    int ret=1;
    if(i < n*m)
        ret = 0;

    CHECK(hipFree(tmp));
    CHECK(hipFree(tmp2));
    CHECK(hipHostFree(h_tmp2));
    CHECK(hipHostFree(h_A));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

    return ret;
}

//Function that computes MoorePenrose pseudoinverse using host libraries.
//The library assumes the input matrix are RowMajor n x m with n >= m
//We use our ColMajor matrices as input which conveniently when read RowMajor with n and m switched are exactly their own transposed, with m >= n.
//This is possible because the pseudoinverse of the transposed is the transposed of the pseudoinverse.
//The n and m switch is performed inside the procedure.
void HostMoorePenroseInverse(double *d_A, int n, int m, double *d_Apinv){

    int nrows = m;
    int ncols = n;
    double *A,*Apinv,*U,*VT,*S,*dummy_array;

    CHECK(hipHostMalloc(&A, n*m*sizeof(double)));
    CHECK(hipHostMalloc(&Apinv, m*n*sizeof(double)));
    CHECK(hipMemcpy(A, d_A, n*m*sizeof(double), hipMemcpyDeviceToHost));

    dummy_array = (double*) malloc(ncols * sizeof(double));
    if(dummy_array == NULL){ printf(" No memory available\n"); exit(0);}

    U = (double *) malloc(nrows * ncols * sizeof(double));
    if(U == NULL){ printf(" No memory available\n"); exit(0);}

    S = (double *) malloc(ncols * sizeof(double));
    if(S == NULL){ printf(" No memory available\n"); exit(0);}

    VT = (double *) malloc(ncols * ncols * sizeof(double));
    if(VT == NULL){ printf(" No memory available\n"); exit(0);}

    int err = Singular_Value_Decomposition(A, nrows, ncols, U, S, VT, dummy_array);

    if(err < 0)
        printf(" Failed to converge\n");

    Singular_Value_Decomposition_Inverse(U, S, VT, 0, nrows, ncols, Apinv);

    CHECK(hipMemcpy(d_Apinv, Apinv, m*n*sizeof(double), hipMemcpyHostToDevice));

    free(dummy_array);
    CHECK(hipHostFree(A));
    CHECK(hipHostFree(Apinv));

}
