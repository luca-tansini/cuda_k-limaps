#include "hip/hip_runtime.h"
#include <float.h>
#include <hipsolver.h>
#include "singular_value_decomposition.h"

#ifndef _COMMON_H
    #include "common.h"
#endif

#ifndef _MATRIX_PRINT_H
    #include "matrixPrint.h"
#endif

#define BLOCK_SIZE 256

/*
Kernel that takes the vector of singular values S (of length m) and produces the pseudo inverse of the diagonal matrix of S. According to the instances, we assume n >= m. Since [n x m] would be the dimension of S diag matrix, and we also have to transpose the diag matrix, its pseudo inverse will be [m x n], with leading dimension m.
The elements on the main diagonal are to be inverted only if non-zero. To determine what is zero we use a threshold based on the machine DBL_EPSILON.
*/
__global__ void calculateDiagPseudoInv(double *S, double *SPseudoInv, int n, int m){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < m){
        if(fabsf(S[idx]) > DBL_EPSILON)
            SPseudoInv[idx*m + idx] = 1/S[idx];
        else
            SPseudoInv[idx*m + idx] = S[idx];
    }
}

/*
This function calcuates the Moore-Penrose inverse matrix of the input matrix A (n*m, with n > m), leaving the result in APseudoInv, assumed preallocated.

The pseudoinverse is computed via SVD.
If SVD(A) = U*S*V^T --> A^+ = V * S^+ * U^T, where S^+ is obtained replacing each non-zero element on the diagonal with its reciprocal and transposing.

The cuSOLVER libraries used to calculate the SVD need the input matrix to be n x m with n >= m.
*/
void MoorePenroseInverse(double *A, int n, int m, double *APseudoInv){

    if(n < m){
        printf("error: n must be >= m! (you can transpose the input matrix and then transpose the result to work with matrices that have less rows than columns)\n");
        return;
    }

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Get dimension needed for the workspace buffer and allocate it
    int bufferDim;
    double *buffer;
    CHECK_CUSOLVER(hipsolverDnDgesvd_bufferSize(cusolverHandle, n, m, &bufferDim));
    CHECK(hipMalloc(&buffer,bufferDim*sizeof(double)));

    //Allocate U,S,V_T
    double *U,*S,*V_T;
    CHECK(hipMalloc(&U, n*n*sizeof(double)));
    CHECK(hipMalloc(&S, m*sizeof(double)));
    CHECK(hipMalloc(&V_T, m*m*sizeof(double)));

    //Calculate SVD with cuSOLVER

    double *Acopy; //we use a copy of A because apparently gesvd destroys input matrix
    CHECK(hipMalloc(&Acopy, n*m*sizeof(double)));
    CHECK(hipMemcpy(Acopy, A, n*m*sizeof(double), hipMemcpyDeviceToDevice));

    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnDgesvd(cusolverHandle, 'A', 'A', n, m, Acopy, n, S, U, n, V_T, m, buffer, bufferDim, NULL, dev_info));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //Calculate S^+
    double *SPseudoInv;
    CHECK(hipMalloc(&SPseudoInv, m*n*sizeof(double)));
    CHECK(hipMemset(SPseudoInv, 0, m*n*sizeof(double)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(m*1.0/BLOCK_SIZE),1,1);
    calculateDiagPseudoInv<<<dimGrid,dimBlock>>>(S, SPseudoInv, n, m);
    CHECK(hipDeviceSynchronize());

    //calculate APseudoInv = V_T^T * S^+ * U^T
    //APseudoInv = V_T^T * S^+
    double alpha=1,beta=0;
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, m, &alpha, V_T, m, SPseudoInv, m, &beta, APseudoInv, m));
    //APseudoInv *= U^T
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, n, &alpha, APseudoInv, m, U, n, &beta, APseudoInv, m));

    //Free memory
    CHECK(hipFree(buffer));
    CHECK(hipFree(U));
    CHECK(hipFree(S));
    CHECK(hipFree(SPseudoInv));
    CHECK(hipFree(V_T));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandle));

}

void TransposedMoorePenroseInverse(double *A, int n, int m, double *APseudoInv){

    double *AT,*APseudoInvT;
    CHECK(hipMalloc(&AT, m*n*sizeof(double)));
    CHECK(hipMalloc(&APseudoInvT, n*m*sizeof(double)));

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    double alpha = 1, beta = 0;

    //Transpose A
    CHECK_CUBLAS(hipblasDgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, A, n, &beta, A, n, AT, m));

    //Call MoorePenroseInverse
    MoorePenroseInverse(AT, m, n, APseudoInvT);

    //Transpose APseudoInvT
    CHECK_CUBLAS(hipblasDgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, APseudoInvT, n, &beta, APseudoInvT, n, APseudoInv, m));

    CHECK(hipFree(AT));
    CHECK(hipFree(APseudoInvT));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

}

void DebugMoorePenroseInverse(double *A, int n, int m, double *APseudoInv){

    if(n < m){
        printf("error: n must be >= m! (you can transpose the input matrix and then transpose the result to work with matrices that have less rows than columns)\n");
        return;
    }

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Get dimension needed for the workspace buffer and allocate it
    int bufferDim;
    double *buffer;
    CHECK_CUSOLVER(hipsolverDnDgesvd_bufferSize(cusolverHandle, n, m, &bufferDim));
    CHECK(hipMalloc(&buffer,bufferDim*sizeof(double)));

    //Allocate U,S,V_T
    double *U,*S,*V_T;
    CHECK(hipMalloc(&U, n*n*sizeof(double)));
    CHECK(hipMalloc(&S, m*sizeof(double)));
    CHECK(hipMalloc(&V_T, m*m*sizeof(double)));

    //Calculate SVD with cuSOLVER

    double *Acopy; //we use a copy of A because apparently gesvd destroys input matrix
    CHECK(hipMalloc(&Acopy, n*m*sizeof(double)));
    CHECK(hipMemcpy(Acopy, A, n*m*sizeof(double), hipMemcpyDeviceToDevice));

    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnDgesvd(cusolverHandle, 'A', 'A', n, m, Acopy, n, S, U, n, V_T, m, buffer, bufferDim, NULL, dev_info));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //Calculate S^+
    double *SPseudoInv;
    CHECK(hipMalloc(&SPseudoInv, m*n*sizeof(double)));
    CHECK(hipMemset(SPseudoInv, 0, m*n*sizeof(double)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(m*1.0/BLOCK_SIZE),1,1);
    calculateDiagPseudoInv<<<dimGrid,dimBlock>>>(S, SPseudoInv, n, m);
    CHECK(hipDeviceSynchronize());

    //*********************************DEBUG************************************

    double *h_U,*h_S,*h_V_T,*h_SPseudoInv;
    CHECK(hipHostMalloc(&h_U, n*n*sizeof(double)));
    CHECK(hipHostMalloc(&h_S, m*sizeof(double)));
    CHECK(hipHostMalloc(&h_V_T, m*m*sizeof(double)));
    CHECK(hipHostMalloc(&h_SPseudoInv, m*n*sizeof(double)));
    CHECK(hipMemcpy(h_U, U, n*n*sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_S, S, m*sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_V_T, V_T, m*m*sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_SPseudoInv, SPseudoInv, m*n*sizeof(double), hipMemcpyDeviceToHost));

    printf("\nU:\n");
    printColumnMajorMatrixForPython(h_U, n, n);

    printf("\nS:\n");
    printColumnMajorMatrixForPython(h_S, 1, m);

    printf("\nVT:\n");
    printColumnMajorMatrixForPython(h_V_T, m, m);

    printf("\nSPseudoInv:\n");
    printColumnMajorMatrixForPython(h_SPseudoInv, m, n);
    printf("\n");

    //******************************END DEBUG***********************************

    //calculate APseudoInv = V_T^T * S^+ * U^T
    //APseudoInv = V_T^T * S^+
    double alpha=1,beta=0;
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, m, &alpha, V_T, m, SPseudoInv, m, &beta, APseudoInv, m));
    //APseudoInv *= U^T
    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, n, &alpha, APseudoInv, m, U, n, &beta, APseudoInv, m));

    //Free memory
    CHECK(hipFree(buffer));
    CHECK(hipFree(U));
    CHECK(hipFree(S));
    CHECK(hipFree(SPseudoInv));
    CHECK(hipFree(V_T));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandle));

}

void TransposeDebugMoorePenroseInverse(double *A, int n, int m, double *APseudoInv){

    double *AT,*APseudoInvT;
    CHECK(hipMalloc(&AT, m*n*sizeof(double)));
    CHECK(hipMalloc(&APseudoInvT, n*m*sizeof(double)));

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    double alpha = 1, beta = 0;

    //Transpose A
    CHECK_CUBLAS(hipblasDgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, A, n, &beta, A, n, AT, m));

    //DEBUG
    double *h_AT;
    CHECK(hipHostMalloc(&h_AT, m*n*sizeof(double)));
    CHECK(hipMemcpy(h_AT, AT, m*n*sizeof(double), hipMemcpyDeviceToHost));
    printf("A^T:\n");
    printColumnMajorMatrixForPython(h_AT,m,n);
    printf("\n");
    //END DEBUG

    //Call MoorePenroseInverse
    DebugMoorePenroseInverse(AT, m, n, APseudoInvT);

    //Transpose APseudoInvT
    CHECK_CUBLAS(hipblasDgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, APseudoInvT, n, &beta, APseudoInvT, n, APseudoInv, m));

    CHECK(hipFree(AT));
    CHECK(hipFree(APseudoInvT));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

}

//The performed check is A * Apinv * A =?= A
int CheckPseudoInverse(double *A, int n, int m, double *Apinv){

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //tmp = A * Apinv
    double *tmp,alpha=1,beta=0;
    CHECK(hipMalloc(&tmp, n*n*sizeof(double)));

    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, m, &alpha, A, n, Apinv, m, &beta, tmp, n));

    //tmp2 = tmp * A
    double *tmp2;
    CHECK(hipMalloc(&tmp2, n*m*sizeof(double)));

    CHECK_CUBLAS(hipblasDgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, n, &alpha, tmp, n, A, n, &beta, tmp2, n));

    //tmp2 =?= A
    double *h_tmp2,*h_A;
    CHECK(hipHostMalloc(&h_tmp2, n*m*sizeof(double)));
    CHECK(hipMemcpy(h_tmp2, tmp2, n*m*sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipHostMalloc(&h_A, n*m*sizeof(double)));
    CHECK(hipMemcpy(h_A, A, n*m*sizeof(double), hipMemcpyDeviceToHost));

    int i;
    for(i=0; i<n*m; i++)
        if(fabs(h_A[i] - h_tmp2[i]) > 1e-4){
            printf("at index %d diff is: %f\n",i, h_A[i] - h_tmp2[i]);
            break;
        }

    int ret=1;
    if(i < n*m)
        ret = 0;

    CHECK(hipFree(tmp));
    CHECK(hipFree(tmp2));
    CHECK(hipHostFree(h_tmp2));
    CHECK(hipHostFree(h_A));
    CHECK_CUBLAS(hipblasDestroy(cublasHandle));

    return ret;
}

//Function that computes MoorePenrose pseudoinverse using host libraries.
//The library assumes the input matrix are RowMajor n x m with n >= m
//We use our ColMajor matrices as input which conveniently when read RowMajor with n and m switched are exactly their own transposed, with m >= n.
//This is possible because the pseudoinverse of the transposed is the transposed of the pseudoinverse.
//The n and m switch is performed inside the procedure.
void HostMoorePenroseInverse(double *d_A, int n, int m, double *d_Apinv){

    int nrows = m;
    int ncols = n;
    double *A,*Apinv,*U,*VT,*S,*dummy_array;

    CHECK(hipHostMalloc(&A, n*m*sizeof(double)));
    CHECK(hipHostMalloc(&Apinv, m*n*sizeof(double)));
    CHECK(hipMemcpy(A, d_A, n*m*sizeof(double), hipMemcpyDeviceToHost));

    dummy_array = (double*) malloc(ncols * sizeof(double));
    if(dummy_array == NULL){ printf(" No memory available\n"); exit(0);}

    U = (double *) malloc(nrows * ncols * sizeof(double));
    if(U == NULL){ printf(" No memory available\n"); exit(0);}

    S = (double *) malloc(ncols * sizeof(double));
    if(S == NULL){ printf(" No memory available\n"); exit(0);}

    VT = (double *) malloc(ncols * ncols * sizeof(double));
    if(VT == NULL){ printf(" No memory available\n"); exit(0);}

    int err = Singular_Value_Decomposition(A, nrows, ncols, U, S, VT, dummy_array);

    if(err < 0)
        printf(" Failed to converge\n");

    Singular_Value_Decomposition_Inverse(U, S, VT, 0, nrows, ncols, Apinv);

    CHECK(hipMemcpy(d_Apinv, Apinv, m*n*sizeof(double), hipMemcpyHostToDevice));

    free(dummy_array);
    CHECK(hipHostFree(A));
    CHECK(hipHostFree(Apinv));

}
