#include "hip/hip_runtime.h"
#include "k-LiMapS.cu"
#include "MoorePenrose.cu"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 256

__global__ void normfill(double *D, int len, hiprandState *states, int seed){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < len){
        hiprand_init(tid*seed+seed, 0, 0, &states[tid]);
        D[tid] = hiprand_uniform_double(&states[tid]);
    }

}

__global__ void divide(double *v, double x, int len){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < len)
        v[tid] /= x;
}

//Function generating the dictionary
//The values are extracted from a normal distribution (mean 0, stdev 1.0)
//Then each column is forced to have norm == 1, dividing each element of the column by the column's norm
void createDict(double *D, int n, int m){

    int blocksperdict = ceil(n*m*1.0/BLOCK_SIZE);

    srand(time(NULL));
    int seed = rand();

    hiprandState *devStates;
    CHECK(hipMalloc((void **)&devStates, blocksperdict*BLOCK_SIZE*sizeof(hiprandState)));

    normfill<<<blocksperdict,BLOCK_SIZE>>>(D, n*m, devStates, seed);
    CHECK(hipDeviceSynchronize());

    double *tmpcol,*partialNormBlocks,norm;
    int blockspercol = ceil(n*1.0/BLOCK_SIZE);
    CHECK(hipMalloc(&tmpcol, blockspercol*BLOCK_SIZE*sizeof(double)));
    CHECK(hipMemset(tmpcol, 0, blockspercol*BLOCK_SIZE*sizeof(double)));
    CHECK(hipHostMalloc(&partialNormBlocks, blockspercol*sizeof(double)));

    for(int i=0; i<m; i++){

        CHECK(hipMemcpy(tmpcol, &D[i*n], n*sizeof(double), hipMemcpyDeviceToDevice));

        //CALCOLA NORMA CON vector2norm
        vector2norm<<<blockspercol,BLOCK_SIZE>>>(tmpcol);
        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy(partialNormBlocks, tmpcol, blockspercol*sizeof(double), hipMemcpyDeviceToHost));
        norm = 0;
        for(int j=0; j<blockspercol; j++)
            norm += partialNormBlocks[j];
        norm = sqrt(norm);

        //CHIAMA KERNEL CHE DIVIDE OGNI ELEMENTO PER LA NORMA
        divide<<<blockspercol,BLOCK_SIZE>>>(&D[i*n], norm, n);
        CHECK(hipDeviceSynchronize());
    }
}

void generateAlpha(double *alpha, int m, int k){

    int blocksperk = ceil(k*1.0/BLOCK_SIZE);

    srand(time(NULL));
    int seed = rand();

    hiprandState *devStates;
    CHECK(hipMalloc(&devStates, blocksperk*BLOCK_SIZE*sizeof(hiprandState)));

    double *d_kvalues,h_kvalues[k];
    CHECK(hipMalloc(&d_kvalues, k*sizeof(double)));

    normfill<<<blocksperk,BLOCK_SIZE>>>(d_kvalues, k, devStates, seed);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(h_kvalues, d_kvalues, k*sizeof(double), hipMemcpyDeviceToHost));

    double h_alpha[m];
    memset(h_alpha, 0, m*sizeof(double));
    for(int i=0; i<k; i++){
        int idx = rand()%m;
        if(h_alpha[idx] != 0)
            i--;
        else
            h_alpha[idx] = h_kvalues[i];
    }

    CHECK(hipMemcpy(alpha, h_alpha, m * sizeof(double), hipMemcpyHostToDevice));

}

int main(int argc, char **argv){

    if(argc != 2){
        printf("usage: noiselessTest <n>\n");
        return 2;
    }

    setbuf(stdout, NULL);

    int n = atoi(argv[1]);
    int m = 2*n;
    int k = n/4;

    //DEBUG
    printf("n:%d\tm:%d\tk:%d\n",n,m,k);
    //END DEBUG

    //CREA DIZIONARIO D
    printf("creating dictionary D...");
    double *D;
    CHECK(hipMalloc(&D, n*m*sizeof(double)));

    createDict(D, n, m);
    printf("done\n");

    //CALCOLA PSEUDOINVERSA DINV
    printf("computing pseudoinverse DINV...");
    double *DINV;
    CHECK(hipMalloc(&DINV, m*n*sizeof(double)));

    HostMoorePenroseInverse(D, n, m, DINV);

    if(!CheckPseudoInverse(D, n, m, DINV))
        printf("Something went wrong with the Moore-Penrose pseudoinverse!\n");

    printf("done\n");

    //GENERA ALPHAOPT
    printf("generating alphaopt...");
    double *alphaopt;
    CHECK(hipMalloc(&alphaopt, m*sizeof(double)));

    generateAlpha(alphaopt, m, k);
    printf("done\n");

    //CALCOLA S = D * alphaopt
    printf("computing S = D * alphaopt...");
    double *s;
    CHECK(hipMalloc(&s, n*sizeof(double)));

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    double cualpha=1,cubeta=0;

    CHECK_CUBLAS(hipblasDgemv(cublasHandle, HIPBLAS_OP_N, n, m, &cualpha, D, n, alphaopt, 1, &cubeta, s, 1));
    printf("done\n");

    //CHIAMA K_LiMapS
    printf("calling K_LiMapS...");
    double *alphalimaps;
    CHECK(hipMalloc(&alphalimaps, m*sizeof(double)));

    devMemK_LiMapS(k, D, n, m, DINV, s, alphalimaps, 1000);
    printf("done\n");

    //CHECK DEL RISULTATO
    double *h_alphalimaps,*h_alphaopt;
    CHECK(hipHostMalloc(&h_alphaopt, m*sizeof(double)));
    CHECK(hipHostMalloc(&h_alphalimaps, m*sizeof(double)));
    CHECK(hipMemcpy(h_alphaopt, alphaopt, m*sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_alphalimaps, alphalimaps, m*sizeof(double), hipMemcpyDeviceToHost));

    int i;
    for(i=0; i<m; i++)
        if(fabs(h_alphaopt[i] - h_alphalimaps[i]) > 1e-3)
            break;
    if(i == m)
        printf("ALL GOOD\n");
    else{
        printf("SOMETHING WENT WRONG!\n");

        printf("alphaopt:\n");
        printHighlightedVector(h_alphaopt, m);
        printf("\n");

        printf("alphalimaps:\n");
        printHighlightedVector(h_alphalimaps, m);
        printf("\n");
    }

    //FREE
    CHECK(hipFree(D));
    CHECK(hipFree(DINV));
    CHECK(hipFree(alphaopt));
    CHECK(hipFree(s));
    CHECK(hipFree(alphalimaps));

    return 0;

}
