#include "hip/hip_runtime.h"
#include "k-LiMapS.cu"
#include "MoorePenrose.cu"

//Function generating the dictionary
void createDict(float *D, int n, int m){

    int i;

    srand(time(NULL));

    for(i=0;i<n*m;i++)
        D[i] = rand()/(float)RAND_MAX;

}

int main(int argc, char **argv){

    if(argc != 2){
        printf("usage: noiselessTest <n>\n");
        return 2;
    }

    int n = atoi(argv[1]);
    int m = 2*n;
    int k = n/4;

    //DEBUG
    printf("n: %d\nm:%d\nk:%d\n",n,m,k);
    //END DEBUG

    //CREA DIZIONARIO D
    float *h_D,*d_D;
    CHECK(hipHostMalloc(&h_D, n*m*sizeof(float)));

    createDict(h_D, n, m);

    CHECK(hipMalloc(&d_D, n*m*sizeof(float)));
    CHECK(hipMemcpy(d_D, h_D, n*m*sizeof(float), hipMemcpyHostToDevice));

    //DEBUG
    printf("D:\n");
    printColumnMajorMatrixForPython(h_D, n, m);
    printf("\n");
    //END DEBUG

    //CALCOLA PSEUDOINVERSA DINV
    float *h_DINV,*d_DINV;
    CHECK(hipMalloc(&d_DINV, m*n*sizeof(float)));

    TransposedMoorePenroseInverse(d_D, n, m, d_DINV);

    if(!CheckPseudoInverse(d_D, n, m, d_DINV)){
        printf("Something went wrong with the Moore-Penrose pseudoinverse!\n");
        return 3;
    }

    CHECK(hipHostMalloc(&h_DINV, m*n*sizeof(float)));
    CHECK(hipMemcpy(h_DINV, d_DINV, m*n*sizeof(float), hipMemcpyDeviceToHost));

    //DEBUG
    printf("DINV:\n");
    printColumnMajorMatrixForPython(h_DINV, m, n);
    printf("\n");
    //END DEBUG

    //GENERA ALPHAOPT
    float *h_alphaopt,*d_alphaopt;
    CHECK(hipHostMalloc(&h_alphaopt, m*sizeof(float)));

    int i,j;
    memset(h_alphaopt, 0, m*sizeof(float));
    for(i=0; i<k; i++){
        j = rand()%m;
        if(h_alphaopt[j] != 0)
            i--;
        else
            h_alphaopt[j] = rand()/(float)RAND_MAX;
    }

    CHECK(hipMalloc(&d_alphaopt, m*sizeof(float)));
    CHECK(hipMemcpy(d_alphaopt, h_alphaopt, m*sizeof(float), hipMemcpyHostToDevice));

    //DEBUG
    printf("alphaopt:\n");
    printColumnMajorMatrixForPython(h_alphaopt, m, 1);
    printf("\n");
    //END DEBUG

    //GENERA S = D * alphaopt
    float *h_s,*d_s;
    CHECK(hipMalloc(&d_s, n*sizeof(float)));

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    float cualpha=1,cubeta=0;

    CHECK_CUBLAS(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, n, m, &cualpha, d_D, n, d_alphaopt, 1, &cubeta, d_s, 1));

    CHECK(hipHostMalloc(&h_s, n*sizeof(float)));
    CHECK(hipMemcpy(h_s, d_s, n*sizeof(float), hipMemcpyDeviceToHost));

    //DEBUG
    printf("s:\n");
    printColumnMajorMatrixForPython(h_s, n, 1);
    printf("\n");
    //END DEBUG

    //CHIAMA K_LiMapS
    float *h_alphalimaps,*d_alphalimaps;
    CHECK(hipMalloc(&d_alphalimaps, m*sizeof(float)));

    devMemK_LiMapS(k, d_D, n, m, d_DINV, d_s, d_alphalimaps, 1000);

    CHECK(hipHostMalloc(&h_alphalimaps, m*sizeof(float)));
    CHECK(hipMemcpy(h_alphalimaps, d_alphalimaps, m*sizeof(float), hipMemcpyDeviceToHost));

    //DEBUG
    printf("alphalimaps:\n");
    printColumnMajorMatrixForPython(h_alphalimaps, m, 1);
    printf("\n");
    //END DEBUG

    //CHECK DEL RISULTATO
    for(i=0; i<m; i++)
        if(fabs(h_alphaopt[i] - h_alphalimaps[i]) > 1e-3)
            break;
    if(i == m)
        printf("ALL GOOD\n");
    else
        printf("SOMETHING WENT WRONG!\n");

    //FREE
    CHECK(hipHostFree(h_D));
    CHECK(hipFree(d_D));
    CHECK(hipHostFree(h_DINV));
    CHECK(hipFree(d_DINV));
    CHECK(hipHostFree(h_alphaopt));
    CHECK(hipFree(d_alphaopt));
    CHECK(hipHostFree(h_s));
    CHECK(hipFree(d_s));
    CHECK(hipHostFree(h_alphalimaps));
    CHECK(hipFree(d_alphalimaps));

    return 0;

}
