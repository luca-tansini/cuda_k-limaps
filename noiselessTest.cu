#include "hip/hip_runtime.h"
#include "k-LiMapS.cu"
#include "vectorUtility.cu"
#include "MoorePenrose.cu"
#include "createDict.cu"

/*
Function calculating MSE: sum((s - D * alphalimaps)^2)/n
*/
double MSE(double *s, double *D, double *alpa, int n, int m){

    int blocks = ceil(n*1.0/BLOCK_SIZE);
    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(BLOCK_SIZE,1,1);
    double *limapsS,*partialMSEBlocks;

    CHECK(hipMalloc(&limapsS, blocks*BLOCK_SIZE*sizeof(double)));
    CHECK(hipMemset(limapsS, 0, blocks*BLOCK_SIZE*sizeof(double)));
    CHECK(hipHostMalloc(&partialMSEBlocks, blocks*sizeof(double)));

    //limapsS = D * alphalimaps
    CHECK_CUBLAS(hipblasDgemv(cublasHandle, HIPBLAS_OP_N, n, m, &cualpha, D, n, alphalimaps, 1, &cubeta, limapsS, 1));

    //limapsS = s - limapsS
    vectorSum<<<dimGrid,dimBlock>>>(1, s, -1, limapsS, limapsS, n);
    CHECK(hipDeviceSynchronize());

    vector2norm<<<dimGrid,dimBlock>>>(d_limapsB);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(partialMSEBlocks, d_limapsB, blocks * sizeof(double), hipMemcpyDeviceToHost));
    double MSE = 0;
    for(j=0; j<blocks; j++)
        MSE += partialMSEBlocks[j];
    MSE /= n;

    CHECK(hipFree(limapsS));
    CHECK(hipHostFree(partialMSEBlocks));

    return MSE;
}

int main(int argc, char **argv){

    if(argc != 2){
        printf("usage: noiselessTest <n>\n");
        return 2;
    }

    setbuf(stdout, NULL);

    int n = atoi(argv[1]);
    int m = 2*n;
    int k = n/4;

    //DEBUG
    printf("n:%d\tm:%d\tk:%d\n",n,m,k);
    //END DEBUG

    //CREA DIZIONARIO D
    printf("creating dictionary D...");
    double *D;
    CHECK(hipMalloc(&D, n*m*sizeof(double)));

    createDict(D, n, m);
    printf("done\n");

    //CALCOLA PSEUDOINVERSA DINV
    printf("computing pseudoinverse DINV...");
    double *DINV;
    CHECK(hipMalloc(&DINV, m*n*sizeof(double)));

    JacobiMoorePenroseInverse(D, n, m, DINV);
    printf("done\n");

    if(!CheckPseudoinverse(D, n, m, DINV))
        printf("Something went wrong with the Moore-Penrose pseudoinverse!\n");

    //GENERA ALPHAOPT
    printf("generating alphaopt...");
    double *alphaopt;
    CHECK(hipMalloc(&alphaopt, m*sizeof(double)));

    generateAlpha(alphaopt, m, k);
    printf("done\n");

    //CALCOLA s = D * alphaopt
    printf("computing s = D * alphaopt...");
    double *s;
    CHECK(hipMalloc(&s, n*sizeof(double)));

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    double cualpha=1,cubeta=0;

    CHECK_CUBLAS(hipblasDgemv(cublasHandle, HIPBLAS_OP_N, n, m, &cualpha, D, n, alphaopt, 1, &cubeta, s, 1));
    printf("done\n");

    //CHIAMA K_LiMapS
    printf("calling K_LiMapS...");
    double *alphalimaps;
    CHECK(hipMalloc(&alphalimaps, m*sizeof(double)));

    devMemK_LiMapS(k, D, n, m, DINV, s, alphalimaps, 1000);
    printf("done\n");

    //CHECK DEL RISULTATO
    double *h_alphalimaps,*h_alphaopt;
    CHECK(hipHostMalloc(&h_alphaopt, m*sizeof(double)));
    CHECK(hipHostMalloc(&h_alphalimaps, m*sizeof(double)));
    CHECK(hipMemcpy(h_alphaopt, alphaopt, m*sizeof(double), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_alphalimaps, alphalimaps, m*sizeof(double), hipMemcpyDeviceToHost));

    int i;
    for(i=0; i<m; i++)
        if(fabs(h_alphaopt[i] - h_alphalimaps[i]) > 1e-3)
            break;
    if(i == m)
        printf("ALL GOOD\n");
    else{
        printf("SOMETHING WENT WRONG!\n");

        printf("alphaopt:\n");
        printHighlightedVector(h_alphaopt, m);
        printf("\n");

        printf("alphalimaps:\n");
        printHighlightedVector(h_alphalimaps, m);
        printf("\n");
    }

    printf("MSE: %f\n", MSE(s,D,alphalimaps,n,m));

    //FREE
    CHECK(hipFree(D));
    CHECK(hipFree(DINV));
    CHECK(hipFree(alphaopt));
    CHECK(hipFree(s));
    CHECK(hipFree(alphalimaps));

    return 0;

}
