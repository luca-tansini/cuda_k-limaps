#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hipblas.h"
#include "common.h"
#include "matrixPrint.h"
#include <hipsolver.h>

int main(int argc, char **argv){

    int n,m,i;
    n = 7;
    m = 3;

    srand(time(NULL));

    float theta[n*m];
    //Fill theta with random values between 0 and 1
    for(i=0; i<n*m; i++)
        theta[i] = rand()/(float)RAND_MAX;

    //DEBUG
    printf("theta:\n");
    printColumnMajorMatrix(theta, n, m);

    printf("theta for Python use:\n");
    printColumnMajorMatrixForPython(theta, n, m);
    //END DEBUG

    float *d_theta,*d_thetaPseudoInv;
    CHECK(hipMalloc(&d_theta, n*m*sizeof(float)));
    CHECK(hipMemcpy(d_theta, theta, n*m*sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc(&d_thetaPseudoInv, m*n*sizeof(float)));

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Get dimension needed for the workspace buffer and allocate it
    int bufferDim;
    CHECK_CUSOLVER(hipsolverDnSgesvd_bufferSize(cusolverHandle, n, m, &bufferDim));
    float *buffer;
    CHECK(hipMalloc(&buffer,bufferDim));

    //Allocate U,S,V_T
    float *U,*S,*V_T;
    CHECK(hipMalloc(&U, n*n*sizeof(float)));
    //S that should be a diagonal matrix is returned as a simple vector instead
    CHECK(hipMalloc(&S, m*sizeof(float)));
    CHECK(hipMalloc(&V_T, m*m*sizeof(float)));

    //Calculate SVD
    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnSgesvd(cusolverHandle, 'A', 'A', n, m, d_theta, n, S, U, n, V_T, m, buffer, bufferDim, NULL, dev_info));

    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));

    //retrieve results
    float *h_U, *h_S, *h_V_T;

    CHECK(hipHostMalloc(&h_U,n*n*sizeof(float)));
    CHECK(hipHostMalloc(&h_S,m*sizeof(float)));
    CHECK(hipHostMalloc(&h_V_T,m*m*sizeof(float)));

    CHECK(hipMemcpy(h_U, U, n*n*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_S, S, m*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_V_T, V_T, m*m*sizeof(float), hipMemcpyDeviceToHost));

    printf("\n\nU (%d*%d):\n\n",n,n);
    printColumnMajorMatrix(h_U,n,n);

    printf("\n\nS (%d):\n\n",m);
    for(i=0;i<m;i++)
        printf("%.3f ", h_S[i]);

    printf("\n\nV_T (%d*%d):\n\n",m,m);
    printColumnMajorMatrix(h_V_T,m,m);

    printf("\n");

    return 0;

}
