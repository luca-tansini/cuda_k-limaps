#include "hip/hip_runtime.h"
#include "k-LiMapS.cu"
#include "MoorePenrose.cu"
#include <sys/time.h>

#ifndef _MATRIX_PRINT_H
    #include "matrixPrint.h"
#endif

double seconds(){
	struct timeval tp;
	struct timezone tzp;
	gettimeofday(&tp, &tzp);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

//Kernel implementing the square sum of a vector (vector is destroyed after computation, with v[i] being the partial sum of block i). The exceeding portion of the vector must be set to 0.
__global__ void squareVectorReduceSum(float *v){

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    v[tid] *= v[tid];

    int step = blockDim.x / 2;
    int idx = threadIdx.x;
    float *p = v + blockDim.x * blockIdx.x;
    while(step > 0){
        if(idx < step)
            p[idx] = p[idx] + p[idx+step];
        step /= 2;
        __syncthreads();
    }
    if(idx == 0)
        v[blockIdx.x] = p[idx];
}

/*
This main tests the k-LiMapS algorithm reading the dictionary, its pseudoinverse and #iter random k-sparse alpha vectors from stdin.
*/
int main(int argc, char **argv){

    int n,m,k,i,j,numIter;
    scanf("%d", &n);
    scanf("%d", &m);
    scanf("%d", &k);
    scanf("%d", &numIter);

    float *theta, *thetaPseudoInv, *alpha, *limapsAlpha, *d_limapsAlpha;
    CHECK(hipHostMalloc(&theta, n*m*sizeof(float)));
    CHECK(hipHostMalloc(&thetaPseudoInv, n*m*sizeof(float)));
    CHECK(hipHostMalloc(&alpha, m*sizeof(float)));
    CHECK(hipHostMalloc(&limapsAlpha, m*sizeof(float)));
    CHECK(hipMalloc(&d_limapsAlpha, m*sizeof(float)));

    //Read theta with random values between 0 and 1
    for(i=0; i<n*m; i++)
        scanf("%f", &theta[i]);
    float *d_theta;
    CHECK(hipMalloc(&d_theta, n*m*sizeof(float)));
    CHECK(hipMemcpy(d_theta, theta, n*m*sizeof(float), hipMemcpyHostToDevice));

    //Read theta Moore-Penrose inverse
    for(i=0; i<m*n; i++)
        scanf("%f", &thetaPseudoInv[i]);
    float *d_thetaPseudoInv;
    CHECK(hipMalloc(&d_thetaPseudoInv, m*n*sizeof(float)));
    CHECK(hipMemcpy(d_thetaPseudoInv, thetaPseudoInv, m*n*sizeof(float), hipMemcpyHostToDevice));

    //Initialize cublas
    float cualpha=1,cubeta=0;
    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Allocate device pointers
    float *d_b, *d_alpha;
    CHECK(hipMalloc(&d_b, n*sizeof(float)));
    CHECK(hipMalloc(&d_alpha, m*sizeof(float)));

    //Allocate MSE temp pointer
    int blocks = ceil(n*1.0/BLOCK_SIZE);
    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(BLOCK_SIZE,1,1);
    float *d_limapsB;
    CHECK(hipMalloc(&d_limapsB, blocks*BLOCK_SIZE*sizeof(float)));
    CHECK(hipMemset(d_limapsB, 0, blocks*BLOCK_SIZE*sizeof(float)));

    float avgMSE=0;
    float *partialMSEBlocks;
    CHECK(hipHostMalloc(&partialMSEBlocks, blocks*sizeof(float)));
    int succ = 0;
    double t1,avgt=0;

    for(int iter = 0; iter < numIter; iter++ ){

        //Read alpha
        for(i=0; i<m; i++)
            scanf("%f", &alpha[i]);
        CHECK(hipMemcpy(d_alpha, alpha, m*sizeof(float), hipMemcpyHostToDevice));

        //Calculate b = theta * alpha
        CHECK_CUBLAS(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, n, m, &cualpha, d_theta, n, d_alpha, 1, &cubeta, d_b, 1));

        /*//DEBUG
        float *b;
        CHECK(hipHostMalloc(&b, n*sizeof(float)));
        CHECK(hipMemcpy(b, d_b, n*sizeof(float), hipMemcpyDeviceToHost));
        printf("\nb:\n");
        printHighlightedVector(b,n);
        //END DEBUG*/

        //Call k_LiMapS
        t1 = seconds();
        devMemK_LiMapS(k, d_theta, n, m, d_thetaPseudoInv, d_b, d_limapsAlpha, 1000);
        avgt += seconds() - t1;

        /*//DEBUG
        float *limapsAlpha;
        CHECK(hipHostMalloc(&limapsAlpha, m * sizeof(float)));
        CHECK(hipMemcpy(limapsAlpha, d_limapsAlpha, m*sizeof(float), hipMemcpyDeviceToHost));
        printf("\nalpha:\n");
        printHighlightedVector(alpha,m);
        printf("\nlimapsAlpha:\n");
        printHighlightedVector(limapsAlpha,m);
        //END DEBUG*/

        //Check result
        CHECK(hipMemcpy(limapsAlpha, d_limapsAlpha, m*sizeof(float), hipMemcpyHostToDevice));
        for(i=0; i<m; i++)
            if(abs(alpha[i] - limapsAlpha[i]) > 1e-4)
                break;
        if(i == m)
            succ++;


        //Calculate MSE: sum((b - theta * limapsAlpha)^2)/n
        CHECK_CUBLAS(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, n, m, &cualpha, d_theta, n, d_limapsAlpha, 1, &cubeta, d_limapsB, 1));

        /*//DEBUG
        float *tmp;
        CHECK(hipHostMalloc(&tmp, blocks*BLOCK_SIZE*sizeof(float)));
        CHECK(hipMemcpy(tmp, d_limapsB, blocks*BLOCK_SIZE*sizeof(float), hipMemcpyDeviceToHost));
        printf("\ntheta * limapsAlpha:\n");
        printHighlightedVector(tmp, blocks*BLOCK_SIZE);
        //END DEBUG*/

        vectorSum<<<dimGrid,dimBlock>>>(1, d_b, -1, d_limapsB, d_limapsB, n);
        CHECK(hipDeviceSynchronize());

        /*//DEBUG
        CHECK(hipMemcpy(tmp, d_limapsB, blocks*BLOCK_SIZE*sizeof(float), hipMemcpyDeviceToHost));
        printf("\nb - theta * limapsAlpha:\n");
        printHighlightedVector(tmp, blocks*BLOCK_SIZE);
        //END DEBUG*/

        squareVectorReduceSum<<<dimGrid,dimBlock>>>(d_limapsB);
        CHECK(hipDeviceSynchronize());

        CHECK(hipMemcpy(partialMSEBlocks, d_limapsB, blocks * sizeof(float), hipMemcpyDeviceToHost));
        float MSE = 0;
        for(j=0; j<blocks; j++)
            MSE += partialMSEBlocks[j];
        avgMSE += MSE/n;
    }

    printf("\nsuccess percentage: %.2f\n",succ*100.0/numIter);
    avgMSE/=numIter;
    printf("\naverage MSE: %.15f\n",avgMSE);
    printf("\naverage k-LiMapS execution time: %.6f\n", avgt/numIter);

    //Free memory
    CHECK(hipHostFree(theta));
    CHECK(hipHostFree(alpha));
    CHECK(hipFree(d_theta));
    CHECK(hipFree(d_alpha));
    CHECK(hipFree(d_thetaPseudoInv));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_limapsB));
    CHECK(hipFree(d_limapsAlpha));
    hipDeviceReset();

    return 0;

}
