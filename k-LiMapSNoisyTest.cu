#include "hip/hip_runtime.h"
#include "k-LiMapS.cu"
#include "vectorUtility.cu"
#include "MoorePenrose.cu"
#include "createDict.cu"

/*
Questo programma esegue i test "noisy" per l'algoritmo k-LiMapS
Il dizionario D e il vettore k-sparso alphaopt vengono generati in GPU a partire da una distribuzione gaussiana.
La pseudoinversa viene calcolata in GPU con l'algoritmo di Jacobi per SVD.
Il segnale sNoisy viene calcolato come D * alphaopt + epsilon, dove epsilon è un vettore generato a partire dalla distribuzione gaussiana e scalato di un fattore pari a 10^-4.
Il programma, con numero di righe della matrice n preso in in input da riga di comando, esegue diversi test per valori di m = n,...,5n e k = 10%n,15%n,...,50%n
Per ogni tripla di valori n,m,k vengono eseguite 50 iterazioni, da cui vengono calcolati alcuni valori come:
    succ%:   una stima di quante volte l'algoritmo k-LiMapS ha prodotto una soluzione approssimata alphalimaps tale che la differenza di ogni elemento tra alphalimaps e alphaopt fosse al più 10^-3
    avgMSE:  la media sulle 50 iterazioni del MeanSquareError tra D*alphaopt e D*alphalimaps
    avgTime: la media sulle 50 iterazioni del tempo di calcolo dell'algoritmo k-LiMapS
Viene generato un nuovo dizionario per ogni valore di m, mentre alphaopt viene estratto ad ogni singola iterazione.
*/
int main(int argc, char **argv){

    if(argc != 2){
        printf("usage: noisyTest <n>\n");
        return 2;
    }

    setbuf(stdout, NULL);

    int n = atoi(argv[1]);

    printf("    n|     m| delta|     k|   rho|  succ%%|      avgMSE      | avgTime |\n");

    //Alloca i puntatori alla memoria deviceS
    double *D,*DINV,*alphaopt,*s,*sNoisy,*epsilon,*alphalimaps,*h_alphalimaps,*h_alphaopt;
    CHECK(hipMalloc(&s, n*sizeof(double)));
    CHECK(hipMalloc(&sNoisy, n*sizeof(double)));
    CHECK(hipMalloc(&epsilon, n*sizeof(double)));

    //Alloca i puntatori per hiprand
    int blocks = ceil(n*1.0/BLOCK_SIZE);
    srand(time(NULL));
    hiprandState *devStates;
    CHECK(hipMalloc(&devStates, n*sizeof(hiprandState)));

    //Crea il cublas handle
    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    double cualpha=1,cubeta=0;

    //Ciclo su m da n a 5n, step n
    for(int m = n; m <= 5*n; m += n){

        //Crea il dizionario
        CHECK(hipMalloc(&D, n*m*sizeof(double)));
        createDict(D, n, m);

        //Calcola la pseudoinversa
        CHECK(hipMalloc(&DINV, m*n*sizeof(double)));
        JacobiMoorePenroseInverse(D, n, m, DINV);
        if(!CheckPseudoinverse(D, n, m, DINV))
            printf("Something went wrong with the Moore-Penrose pseudoinverse!\n");

        //Alloca alphaopt e alphalimaps
        CHECK(hipMalloc(&alphaopt, m*sizeof(double)));
        CHECK(hipMalloc(&alphalimaps, m*sizeof(double)));
        CHECK(hipHostMalloc(&h_alphaopt, m*sizeof(double)));
        CHECK(hipHostMalloc(&h_alphalimaps, m*sizeof(double)));

        //Ciclo su k dal 10% di n al 50%, step 5%
        for(int l = 10; l<=50; l+=5){
            int k = n*l/100.0;

            int iters;
            int succ = 0;
            double avgMSE = 0;
            double avgTime = 0;

            //n, m, delta, k, rho
            printf("%5d| %5d| %5.2f| %5d| %5.2f| ", n, m, n*1.0/m, k, k*1.0/n);

            for(iters=0; iters<50; iters++){

                //Genera alphaopt
                generateAlpha(alphaopt, m, k);

                //Calcola sNoisy = D * alphaopt + epsilon (salvando s per il calcolo del MSE)
                CHECK_CUBLAS(hipblasDgemv(cublasHandle, HIPBLAS_OP_N, n, m, &cualpha, D, n, alphaopt, 1, &cubeta, s, 1));
                normfill<<<blocks,BLOCK_SIZE>>>(epsilon, n, devStates, rand());
                hipDeviceSynchronize();
                //Scala epsilon di un fattore 10^-4
                divide<<<blocks,BLOCK_SIZE>>>(epsilon, 10000, n);
                hipDeviceSynchronize();
                vectorSum<<<blocks,BLOCK_SIZE>>>(1, s, 1, epsilon, sNoisy, n);

                //Chiama K_LiMapS
                double t=seconds();
                k_LiMapS(k, D, n, m, DINV, sNoisy, alphalimaps, 1000);
                avgTime += seconds() - t;

                //Check del risultato (succ%)
                CHECK(hipMemcpy(h_alphaopt, alphaopt, m*sizeof(double), hipMemcpyDeviceToHost));
                CHECK(hipMemcpy(h_alphalimaps, alphalimaps, m*sizeof(double), hipMemcpyDeviceToHost));

                int i;
                for(i=0; i<m; i++)
                    if(fabs(h_alphaopt[i] - h_alphalimaps[i]) > 1e-3)
                        break;
                if(i == m)
                    succ++;

                //Calcola MSE
                avgMSE += MSE(s,D,alphalimaps,n,m);

            }

            avgMSE  /= iters;
            avgTime /= iters;

            //succ, avgMSE, avgTime
            printf("%6.2f| %17.15f| %8.6f|\n", succ*100.0/50, avgMSE, avgTime);

        }

        CHECK(hipFree(D));
        CHECK(hipFree(DINV));
        CHECK(hipFree(alphaopt));
        CHECK(hipFree(alphalimaps));
        CHECK(hipHostFree(h_alphaopt));
        CHECK(hipHostFree(h_alphalimaps));

    }

    CHECK(hipFree(s));
    CHECK(hipFree(epsilon));
    CHECK(hipFree(sNoisy));

    return 0;

}
