#include "hip/hip_runtime.h"
#include "k-LiMapS.cu"
#include "MoorePenrose.cu"

/*
This main tests the k-LiMapS algorithm generating the dictionary and its pseudoinverse in GPU.
At the moment MoorePenrose.cu has proven wrong (due to the limitation of the cuSOLVER libraries)
*/
int main(int argc, char **argv){

    if(argc != 3){
        printf("usage: simpleTest n k\n");
        exit(2);
    }

    int n,k,m,i,j;
    n = atoi(argv[1]);
    k = atoi(argv[2]);
    m = n*k;

    srand(time(NULL));

    //Fill theta with random values between 0 and 1
    float *theta;
    CHECK(hipHostMalloc(&theta, n*m*sizeof(float)));
    for(i=0; i<n*m; i++)
        theta[i] = rand()/(float)RAND_MAX;

    //Fill optimal solution alpha with k random values
    float *alpha;
    CHECK(hipHostMalloc(&alpha, m*sizeof(float)));
    memset(alpha,0,m*sizeof(float));
    for(i=0; i<k; i++){
        j = rand()%m;
        if(alpha[j] != 0)
            i--;
        else
            alpha[j] = rand()/(float)RAND_MAX;
    }

    //calculate theta Moore-Penrose inverse
    float *d_theta,*d_thetaPseudoInv;
    CHECK(hipMalloc(&d_theta, n*m*sizeof(float)));
    CHECK(hipMemcpy(d_theta, theta, n*m*sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc(&d_thetaPseudoInv, m*n*sizeof(float)));
    MoorePenroseInverse(d_theta, n, m, d_thetaPseudoInv);

    float *thetaPseudoInv;
    CHECK(hipHostMalloc(&thetaPseudoInv, m*n*sizeof(float)));
    CHECK(hipMemcpy(thetaPseudoInv, d_thetaPseudoInv, m*n*sizeof(float), hipMemcpyDeviceToHost));

    //Calculate b = theta * alpha
    float *d_b,*d_alpha,cualpha=1,cubeta=0;
    hipblasHandle_t cublasHandle;

    CHECK(hipMalloc(&d_b, n*sizeof(float)));
    CHECK(hipMalloc(&d_alpha, m*sizeof(float)));
    CHECK(hipMemcpy(d_alpha, alpha, m*sizeof(float), hipMemcpyHostToDevice));

    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    CHECK_CUBLAS(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, n, m, &cualpha, d_theta, n, d_alpha, 1, &cubeta, d_b, 1));

    //call k_LiMapS

    //For compatibility with future use, k_LiMapS parameters are supposed to be host memory pointers, so we need to transfer MoorePenroseInverse result and d_b into host memory
    float *b, *limapsAlpha;
    CHECK(hipHostMalloc(&b, n*sizeof(float)));
    CHECK(hipMemcpy(b, d_b, n*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipHostMalloc(&limapsAlpha, m*sizeof(float)));

    k_LiMapS(k, theta, n, m, thetaPseudoInv, d_b, limapsAlpha, 10);

    //Check result
    for(i=0; i<m; i++)
        if(abs(alpha[i] - limapsAlpha[i]) > 1e-4)
            break;

    if(i < m){
        printf("NOPE\n");
        printf("\nalpha:\n");
        printHighlightedVector(alpha, m);

        printf("\nlimapsAlpha:\n");
        printHighlightedVector(limapsAlpha, m);
    }
    else
        printf("OK!\n");

    //Free memory
    CHECK(hipHostFree(theta));
    CHECK(hipHostFree(alpha));
    CHECK(hipHostFree(limapsAlpha));
    CHECK(hipHostFree(thetaPseudoInv));
    CHECK(hipFree(d_theta));
    CHECK(hipFree(d_alpha));
    CHECK(hipFree(d_thetaPseudoInv));
    CHECK(hipFree(d_b));
    hipDeviceReset();

    return 0;

}
