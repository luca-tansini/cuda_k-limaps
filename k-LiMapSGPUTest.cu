#include "hip/hip_runtime.h"
#include "k-LiMapS.cu"
#include "MoorePenrose.cu"

/*
This main tests the k-LiMapS algorithm generating the dictionary and its pseudoinverse in GPU.
*/
int main(int argc, char **argv){

    if(argc != 2){
        printf("usage: k-LiMapSGPUTest n\n");
        exit(2);
    }

    int n,i,j;
    n = atoi(argv[1]);

    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));
    float cualpha = 1, cubeta = 0;

    float *d_b,*b;
    CHECK(hipMalloc(&d_b, n*sizeof(float)));
    CHECK(hipHostMalloc(&b, n*sizeof(float)));

    for(int m = n; m <= 5*n; m+=n){

        printf("m = %d\n", m);

        //Generate dictionary theta with uniform random values between 0 and 1
        srand(time(NULL));
        float *theta;
        CHECK(hipHostMalloc(&theta, n*m*sizeof(float)));
        for(i=0; i<n*m; i++)
            theta[i] = rand()/(float)RAND_MAX;

        //calculate theta Moore-Penrose inverse
        float *d_theta,*d_thetaPseudoInv;
        CHECK(hipMalloc(&d_theta, n*m*sizeof(float)));
        CHECK(hipMemcpy(d_theta, theta, n*m*sizeof(float), hipMemcpyHostToDevice));
        CHECK(hipMalloc(&d_thetaPseudoInv, m*n*sizeof(float)));
        TransposedMoorePenroseInverse(d_theta, n, m, d_thetaPseudoInv);
        if(!CheckPseudoInverse(d_theta, n, m, d_thetaPseudoInv)){
            printf("something went wrong with the pseudoinverse!\n");
            return -2;
        }


        //transfer d_thetaPseudoInv into host memory
        float *thetaPseudoInv;
        CHECK(hipHostMalloc(&thetaPseudoInv, m*n*sizeof(float)));
        CHECK(hipMemcpy(thetaPseudoInv, d_thetaPseudoInv, m*n*sizeof(float), hipMemcpyDeviceToHost));

        //Allocate host and device alpha vectors
        float *alpha, *d_alpha, *limapsAlpha;
        CHECK(hipHostMalloc(&alpha, m*sizeof(float)));
        CHECK(hipMalloc(&d_alpha, m*sizeof(float)));
        CHECK(hipHostMalloc(&limapsAlpha, m*sizeof(float)));

        for(int k = ceil(n/10); k <= ceil(n/2); k+= ceil(n/10)){

            int succ = 0;

            for(int iter= 0; iter < 100; iter++){

                //Fill optimal solution alpha with k random values
                memset(alpha,0,m*sizeof(float));
                for(i=0; i<k; i++){
                    j = rand()%m;
                    if(alpha[j] != 0)
                        i--;
                    else
                        alpha[j] = rand()/(float)RAND_MAX;
                }

                CHECK(hipMemcpy(d_alpha, alpha, m*sizeof(float), hipMemcpyHostToDevice));

                //Calculate b = theta * alpha
                CHECK_CUBLAS(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, n, m, &cualpha, d_theta, n, d_alpha, 1, &cubeta, d_b, 1));
                CHECK(hipMemcpy(b, d_b, n*sizeof(float), hipMemcpyDeviceToHost));

                //call k_LiMapS
                k_LiMapS(k, theta, n, m, thetaPseudoInv, b, limapsAlpha, 1000);

                //Check result
                for(i=0; i<m; i++)
                    if(abs(alpha[i] - limapsAlpha[i]) > 1e-4)
                        break;
                if(i == m)
                    succ += 0;
            }

            printf("    k = %d --> %d%%\n", k, succ);

        }

        CHECK(hipFree(d_theta));
        CHECK(hipFree(d_thetaPseudoInv));
        CHECK(hipHostFree(theta));
        CHECK(hipHostFree(thetaPseudoInv));
        CHECK(hipFree(d_alpha));
        CHECK(hipHostFree(alpha));
        CHECK(hipHostFree(limapsAlpha));

    }

    hipDeviceReset();
    return 0;

}
