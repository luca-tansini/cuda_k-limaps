#include "hip/hip_runtime.h"
#include "k-LiMapS.cu"
#include "matrixPrint.h"
#include <float.h>
#include <hipsolver.h>

/*
Kernel that takes the vector of singular values S and produces the pseudo inverse of the diagonal matrix of S. According to the instances, we assume n < m. Since [n x m] would be the dimension of S diag matrix, and we also have to transpose the diag matrix, its pseudo inverse will be [m x n], with leading dimension m.
The elemnts on the main diagonal are to be inverted only if non-zero. To determine what is zero we use a threshold based on the machine FLT_EPSILON.
*/
__global__ void calculateDiagPseudoInv(float *S, float *SPseudoInv, int n, int m){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n)
        if(fabsf(S[idx]) > FLT_EPSILON)
            SPseudoInv[idx*m + idx] = 1/S[idx];
        else
            SPseudoInv[idx*m + idx] = S[idx];
}

/*
This function calcuates the Moore-Penrose inverse matrix of the input matrix A (n*m, with m > n), leaving the result in apseudoinv, assumed preallocated.

The pseudoinverse is computed via SVD.
If SVD(A) = U*S*V^T --> A^+ = V * S^+ * U^T, where S^+ is obtained replacing each non-zero element on the diagonal with its reciprocal.

The cuSOLVER libraries used to calculate the SVD need the input matrix to be n x m with n < m, so we need to transpose our matrix.

According to math
being SVD(A) = U * S * V_T,
being SVD(A^T) = U1 * S1 * V_T1,
we have that
U == V_T1^T (at least in absolute values)
S == S1
V_T == U1^T (at least in absolute values)
A == U * S * V_T == V_T1^T * S1 * U1^T == A

So we can compute A^+ as
U1^T * S1^+ * V_T1

*/
void MoorePenroseInverse(float *A, int n, int m, float *Apseudoinv){

    hipsolverHandle_t cusolverHandle;
    hipblasHandle_t cublasHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //transpose matrix A
    float alpha=1,beta=0;
    CHECK_CUBLAS(hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &alpha, A, m, &beta, A, n, Apseudoinv, n));

    //Get dimension needed for the workspace buffer and allocate it
    int bufferDim;
    float *buffer;
    CHECK_CUSOLVER(hipsolverDnSgesvd_bufferSize(cusolverHandle, m, n, &bufferDim));
    CHECK(hipMalloc(&buffer,bufferDim));

    //Allocate U1,S1,V_T1
    float *U1,*S1,*V_T1;
    CHECK(hipMalloc(&U1, m*m*sizeof(float)));
    CHECK(hipMalloc(&S1, n*sizeof(float)));
    CHECK(hipMalloc(&V_T1, n*n*sizeof(float)));

    //Calculate SVD with cuSOLVER
    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnSgesvd(cusolverHandle, 'A', 'A', m, n, Apseudoinv, m, S1, U1, m, V_T1, n, buffer, bufferDim, NULL, dev_info));
    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info != 0)
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    /*//DEBUG PRINT *******************************
    float *h_S1;

    CHECK(hipHostMalloc(&h_S1,n*sizeof(float)));
    CHECK(hipMemcpy(h_S1, S1, n*sizeof(float), hipMemcpyDeviceToHost));

    printf("\n\nS [%d]:\n\n",n);
    for(int i=0;i<n;i++)
        printf("%.3f ", h_S1[i]);
    printf("\n");

    hipHostFree(h_S1);
    //END DEBUG ********************************/

    //Calculate S1^+
    float *S1PseudoInv;
    CHECK(hipMalloc(&S1PseudoInv, m*n*sizeof(float)));
    CHECK(hipMemset(S1PseudoInv, 0, m*n*sizeof(float)));

    dim3 dimBlock(BLOCK_SIZE,1,1);
    dim3 dimGrid(ceil(n*1.0/BLOCK_SIZE),1,1);
    calculateDiagPseudoInv<<<dimGrid,dimBlock>>>(S1, S1PseudoInv, n, m);
    CHECK(hipDeviceSynchronize());

    /*//DEBUG PRINT *******************************
    float *h_S1PseudoInv;

    CHECK(hipHostMalloc(&h_S1PseudoInv, m*n*sizeof(float)));
    CHECK(hipMemcpy(h_S1PseudoInv, S1PseudoInv, m*n*sizeof(float), hipMemcpyDeviceToHost));

    printf("\n\nS1 pseudoinverse [%dx%d]:\n\n", m, n);
    printColumnMajorMatrix(h_S1PseudoInv, m, n);
    printf("\n");

    hipHostFree(h_S1PseudoInv);
    //END DEBUG ********************************/

    //calculate U1^T * S1^+ * V_T1

    //Free memory
    CHECK(hipFree(buffer));
    CHECK(hipFree(U1));
    CHECK(hipFree(S1));
    CHECK(hipFree(V_T1));

}

int main(int argc, char **argv){

    if(argc != 3){
        printf("usage: simpleTest n k\n");
        exit(2);
    }

    int n,k,m,i,j;
    n = atoi(argv[1]);
    k = atoi(argv[2]);
    m = n*k;

    srand(time(NULL));

    //Fill theta with random values between 0 and 1
    float *theta;
    CHECK(hipHostMalloc(&theta, n*m*sizeof(float)));
    for(i=0; i<n*m; i++)
        theta[i] = rand()/(float)RAND_MAX;


    //Fill optimal solution alpha with k random values
    float *alpha;
    CHECK(hipHostMalloc(&alpha, m*sizeof(float)));
    memset(alpha,0,m*sizeof(float));
    for(i=0; i<k; i++){
        j = rand()%m;
        if(alpha[j] != 0)
            i--;
        else
            alpha[j] = rand()/(float)RAND_MAX;
    }

    /*//DEBUG PRINT
    printf("theta:\n");
    printColumnMajorMatrix(theta, n, m);

    printf("theta for Python use:\n");
    printColumnMajorMatrixForPython(theta, n, m);
    //END DEBUG*/

    //calculate theta Moore-Penrose inverse
    float *d_theta,*d_thetaPseudoInv;
    CHECK(hipMalloc(&d_theta, n*m*sizeof(float)));
    CHECK(hipMemcpy(d_theta, theta, n*m*sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc(&d_thetaPseudoInv, m*n*sizeof(float)));
    MoorePenroseInverse(d_theta, n, m, d_thetaPseudoInv);

    //Calbulate b = theta * alpha

    //call k_LiMapS

    //Check result

    //Free memory
    CHECK(hipHostFree(theta));
    CHECK(hipHostFree(alpha));
    CHECK(hipFree(d_theta));

    return 0;

}
