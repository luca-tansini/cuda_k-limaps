#include "hip/hip_runtime.h"
#include "k-LiMapS.cu"
#include "MoorePenrose.cu"
#ifndef _MATRIX_PRINT_H
    #include "matrixPrint.h"
#endif

/*
This main tests the k-LiMapS algorithm generating the dictionary and its pseudoinverse in GPU.
*/
int main(int argc, char **argv){

    if(argc != 5){
        printf("usage: simpleTest n m k numIter\n");
        exit(-1);
    }

    int n,m,k,i,j,numIter;
    n = atoi(argv[1]);
    m = atoi(argv[2]);
    k = atoi(argv[3]);
    numIter = atoi(argv[4]);

    srand(time(NULL));

    float *theta, *alpha, *b, *limapsAlpha;
    CHECK(hipHostMalloc(&theta, n*m*sizeof(float)));
    CHECK(hipHostMalloc(&alpha, m*sizeof(float)));
    CHECK(hipHostMalloc(&b, n*sizeof(float)));
    CHECK(hipHostMalloc(&limapsAlpha, m*sizeof(float)));

    //Fill theta with random values between 0 and 1
    for(i=0; i<n*m; i++)
        theta[i] = rand()/(float)RAND_MAX;

    //calculate theta Moore-Penrose inverse
    float *d_theta,*d_thetaPseudoInv;
    CHECK(hipMalloc(&d_theta, n*m*sizeof(float)));
    CHECK(hipMemcpy(d_theta, theta, n*m*sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc(&d_thetaPseudoInv, m*n*sizeof(float)));
    TransposedMoorePenroseInverse(d_theta, n, m, d_thetaPseudoInv);
    if(!CheckPseudoInverse(d_theta, n, m, d_thetaPseudoInv)){
        printf("something went wrong with the pseudoinverse!\n");
        return -2;
    }

    float *thetaPseudoInv;
    CHECK(hipHostMalloc(&thetaPseudoInv, m*n*sizeof(float)));
    CHECK(hipMemcpy(thetaPseudoInv, d_thetaPseudoInv, m*n*sizeof(float), hipMemcpyDeviceToHost));

    //Initialize cublas
    float cualpha=1,cubeta=0;
    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //Allocate device pointers
    float *d_b, *d_alpha;
    CHECK(hipMalloc(&d_b, n*sizeof(float)));
    CHECK(hipMalloc(&d_alpha, m*sizeof(float)));

    int succ = 0;
    for(int iter = 0; iter < numIter; iter++ ){

        //Fill optimal solution alpha with k random values
        memset(alpha,0,m*sizeof(float));
        for(i=0; i<k; i++){
            j = rand()%m;
            if(alpha[j] != 0)
                i--;
            else
                alpha[j] = rand()/(float)RAND_MAX;
        }

        CHECK(hipMemcpy(d_alpha, alpha, m*sizeof(float), hipMemcpyHostToDevice));

        //Calculate b = theta * alpha
        CHECK_CUBLAS(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, n, m, &cualpha, d_theta, n, d_alpha, 1, &cubeta, d_b, 1));
        CHECK(hipMemcpy(b, d_b, n*sizeof(float), hipMemcpyDeviceToHost));

        //call k_LiMapS
        k_LiMapS(k, theta, n, m, thetaPseudoInv, b, limapsAlpha, 1000);

        //Check result
        for(i=0; i<m; i++)
            if(abs(alpha[i] - limapsAlpha[i]) > 1e-4)
                break;

        if(i == m)
            succ++;

    }

    printf("%.2f%%\n",100.0*succ/numIter);

    //Free memory
    CHECK(hipHostFree(theta));
    CHECK(hipHostFree(alpha));
    CHECK(hipHostFree(limapsAlpha));
    CHECK(hipHostFree(thetaPseudoInv));
    CHECK(hipFree(d_theta));
    CHECK(hipFree(d_alpha));
    CHECK(hipFree(d_thetaPseudoInv));
    CHECK(hipFree(d_b));
    hipDeviceReset();

    return 0;

}
