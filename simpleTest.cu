#include "k-LiMapS.cu"
#include "matrixPrint.h"
#include <hipsolver.h>

/*
This function calcuates the Moore-Penrose inverse matrix of the input matrix A (n*m, with m > n), leaving the result in apseudoinv, assumed preallocated.
The pseudoinverse is computed via SVD.
If SVD(A) = U*S*V^T --> A^+ = V * S^+ * U^T, where S^+ is obtained replacing each non-zero element on the diagonal with its reciprocal.
The cuSOLVER libraries used to calculate the SVD need the input matrix to be n x m with n < m, so we need to transpose our matrix
*/
void MoorePenroseInverse(float *A, int n, int m, float *Apseudoinv){

    //Calculate theta SVD via cuSOLVER api
    hipsolverHandle_t cusolverHandle;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandle));
    hipblasHandle_t cublasHandle;
    CHECK_CUBLAS(hipblasCreate(&cublasHandle));

    //transpose matrix A
    float alpha=1,beta=0;
    CHECK_CUBLAS(hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &alpha, A, m, &beta, A, n, Apseudoinv, n)); //non so se mettere n o m come ultimo parametro

    //Get dimension needed for the workspace buffer and allocate it
    int bufferDim;
    CHECK_CUSOLVER(hipsolverDnSgesvd_bufferSize(cusolverHandle, m, n, &bufferDim));
    float *buffer;
    CHECK(hipMalloc(&buffer,bufferDim));

    //Remembering A was transposed we have that SVD --> A[m*n] = U[m*m] * S[m*n diag] * V_T[n*n]
    //Allocate U,S,V_T
    float *U,*S,*V_T;
    CHECK(hipMalloc(&U, m*m*sizeof(float)));
    //S that should be a diagonal matrix is returned as a simple vector instead
    CHECK(hipMalloc(&S, n*sizeof(float)));
    CHECK(hipMalloc(&V_T, n*n*sizeof(float)));

    //Calculate SVD (of A^T, we will have to do some considerations on our results)
    int *dev_info, h_dev_info;
    CHECK(hipMalloc(&dev_info, sizeof(int)));
    CHECK_CUSOLVER(hipsolverDnSgesvd(cusolverHandle, 'A', 'A', m, n, Apseudoinv, m, S, U, m, V_T, n, buffer, bufferDim, NULL, dev_info));

    CHECK(hipMemcpy(&h_dev_info, dev_info, sizeof(int), hipMemcpyDeviceToHost));
    if(h_dev_info == 0)
        printf("Success!\n");
    else
        printf("Something went wrong (dev_info=%d)\n", h_dev_info);

    //DEBUG PRINT
    //retrieve results
    float *h_U, *h_S, *h_V_T;

    CHECK(hipHostMalloc(&h_U,m*m*sizeof(float)));
    CHECK(hipHostMalloc(&h_S,n*sizeof(float)));
    CHECK(hipHostMalloc(&h_V_T,n*n*sizeof(float)));

    CHECK(hipMemcpy(h_U, U, m*m*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_S, S, n*sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_V_T, V_T, n*n*sizeof(float), hipMemcpyDeviceToHost));

    printf("\n\nU (%d*%d):\n\n",m,m);
    printColumnMajorMatrix(h_U,m,m);

    printf("\n\nS (%d):\n\n",n);
    for(int i=0;i<n;i++)
        printf("%.3f ", h_S[i]);

    printf("\n\nV_T (%d*%d):\n\n",n,n);
    printColumnMajorMatrix(h_V_T, n, n);

    printf("\n");
    //END DEBUG

}

int main(int argc, char **argv){

    if(argc != 3){
        printf("usage: simpleTest n k\n");
        exit(2);
    }

    int n,k,m,i,j;
    n = atoi(argv[1]);
    k = atoi(argv[2]);
    m = n*k;

    srand(time(NULL));

    float theta[n*m];
    //Fill theta with random values between 0 and 1
    for(i=0; i<n*m; i++)
        theta[i] = rand()/(float)RAND_MAX;

    float alpha[m];
    //Fill optimal solution alpha with k random values
    memset(alpha,0,m*sizeof(float));
    for(i=0; i<k; i++){
        j = rand()%m;
        if(alpha[j] != 0)
            i--;
        else
            alpha[j] = rand()/(float)RAND_MAX;
    }

    //DEBUG
    printf("theta:\n");
    printColumnMajorMatrix(theta, n, m);

    printf("theta for Python use:\n");
    printColumnMajorMatrixForPython(theta, n, m);
    //END DEBUG

    //MoorePenroseInverse
    float *d_theta,*d_thetaPseudoInv;
    CHECK(hipMalloc(&d_theta, n*m*sizeof(float)));
    CHECK(hipMemcpy(d_theta, theta, n*m*sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMalloc(&d_thetaPseudoInv, m*n*sizeof(float)));
    MoorePenroseInverse(d_theta, n, m, d_thetaPseudoInv);

    //Calbulate b = theta * alpha

    //call k_LiMapS

    //Check result

    return 0;

}
